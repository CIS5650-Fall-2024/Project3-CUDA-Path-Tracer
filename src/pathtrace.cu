#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

#define STREAMCOMPACT

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
    thrust::default_random_engine
    makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

// Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4 *pbo, glm::ivec2 resolution, int iter, glm::vec3 *image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::vec3 color = pix / static_cast<float>(iter);
        color = 255.f * color / (color + 1.f);

        glm::ivec3 icolor = color;

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = icolor.x;
        pbo[index].y = icolor.y;
        pbo[index].z = icolor.z;
    }
}

static Scene *hst_scene = NULL;
static GuiDataContainer *guiData = NULL;
static glm::vec3 *dev_image = NULL;
static Geom *dev_geoms = NULL;
static Material *dev_materials = NULL;
static PathSegment *dev_paths = NULL;
static ShadeableIntersection *dev_intersections = NULL;

static Mesh *dev_meshes = NULL;
static int *dev_indices = NULL;
static glm::vec3 *dev_points = NULL;
static glm::vec2 *dev_uvs = NULL;

static std::vector<hipArray_t> texArrays;
static std::vector<hipTextureObject_t> texObjects;

void InitDataContainer(GuiDataContainer *imGuiData)
{
    guiData = imGuiData;
}

void textureInit(const std::vector<TextureData> &textures, std::vector<Material> &materials)
{
    assert(texObjects.empty());
    texObjects.reserve(textures.size());
    
    assert(texArrays.empty());
    texArrays.resize(textures.size());

    hipError_t hipError_t;
    for (size_t i = 0; i < textures.size(); i++)
    {
        const TextureData &texture = textures[i];
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
        hipError_t = hipMallocArray(&texArrays[i], &channelDesc, texture.dimensions.x, texture.dimensions.y);
        if (hipError_t != hipSuccess) {
            std::cerr << "Malloc array error" << hipGetErrorString(hipError_t) << std::endl;
        }

        hipError_t = hipMemcpy2DToArray(texArrays[i], 0, 0, texture.data.data(), texture.dimensions.x * sizeof(glm::vec4), texture.dimensions.x * sizeof(glm::vec4), texture.dimensions.y, hipMemcpyHostToDevice);
        if (hipError_t != hipSuccess) {
            std::cerr << "Memcpy2dToArray error" << hipGetErrorString(hipError_t) << std::endl;
        }

        hipResourceDesc resourceDesc;
        resourceDesc.resType = hipResourceTypeArray;
        resourceDesc.res.array.array = texArrays[i];

        hipTextureDesc textureDesc{
            .addressMode = {hipAddressModeClamp, hipAddressModeClamp, hipAddressModeClamp},
            .filterMode = hipFilterModePoint,
            .readMode = hipReadModeElementType,
            .sRGB = 0,
            .normalizedCoords = 1
        };


        hipTextureObject_t texObject;
        hipError_t = hipCreateTextureObject(&texObject, &resourceDesc, &textureDesc, nullptr);
        if (hipError_t != hipSuccess) {
            std::cerr << "hipCreateTextureObject error" << hipGetErrorString(hipError_t) << std::endl;
        }

        for (auto& material : materials) {
            int32_t albedoIndex = material.albedo.negSuccTexInd;
            if (albedoIndex < 0 && -albedoIndex == i + 1) {
                material.albedo.textureHandle = texObject;
            }
        }
    }
}

void textureFree() {
    for (auto tex : texObjects) {
        hipDestroyTextureObject(tex);
    }
    for (auto arr : texArrays) {
        hipFreeArray(arr);
    }
    texObjects.clear();
    texArrays.clear();
}

void pathtraceInit(Scene *scene)
{
    hst_scene = scene;

    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    textureInit(hst_scene->texes, hst_scene->materials);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_meshes, scene->meshes.size() * sizeof(Mesh));
    hipMemcpy(dev_meshes, scene->meshes.data(), scene->meshes.size() * sizeof(Mesh), hipMemcpyHostToDevice);

    hipMalloc(&dev_indices, scene->indices.size() * sizeof(int));
    hipMemcpy(dev_indices, scene->indices.data(), scene->indices.size() * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&dev_points, scene->positions.size() * sizeof(glm::vec3));
    hipMemcpy(dev_points, scene->positions.data(), scene->positions.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);

    hipMalloc(&dev_uvs, scene->uvs.size() * sizeof(glm::vec2));
    hipMemcpy(dev_uvs, scene->uvs.data(), scene->uvs.size() * sizeof(glm::vec2), hipMemcpyHostToDevice);
    
    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    hipFree(dev_meshes);
    hipFree(dev_indices);
    hipFree(dev_points);
    hipFree(dev_uvs);
    textureFree();
    checkCUDAError("pathtraceFree");
}

/**
 * Generate PathSegments with rays from the camera through the screen into the
 * scene, which is the first bounce of rays.
 *
 * Antialiasing - add rays for sub-pixel sampling
 * motion blur - jitter rays "in time"
 * lens effect - jitter ray origin positions based on a lens
 */
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment *pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (x < cam.resolution.x && y < cam.resolution.y)
    {
        int index = x + (y * cam.resolution.x);
        PathSegment &segment = pathSegments[index];

        Ray ray;
        ray.origin = cam.position;
        segment.throughput = glm::vec3(1.0f, 1.0f, 1.0f);
        segment.radiance = glm::vec3();

        auto rng = makeSeededRandomEngine(iter, index, -1);
        thrust::uniform_real_distribution<float> u01(0, 1);

        ray.direction = glm::normalize(cam.view - cam.right * cam.pixelLength.x * (static_cast<float>(x) + u01(rng) - static_cast<float>(cam.resolution.x) * 0.5f) - cam.up * cam.pixelLength.y * (static_cast<float>(y) + u01(rng) - static_cast<float>(cam.resolution.y) * 0.5f));

        if (cam.lensSize == 0)
        {
            segment.ray = ray;
        }
        else
        {
            glm::vec2 diskPoint = calculateRandomPointOnDisk(rng) * cam.pixelLength * glm::vec2(cam.resolution);
            segment.ray.origin = cam.position + cam.lensSize * (diskPoint.x * cam.right + diskPoint.y * cam.up);

            glm::vec3 focalPoint = cam.position + cam.focalDist * ray.direction;
            segment.ray.direction = glm::normalize(focalPoint - segment.ray.origin);
        }
        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth,
    int num_paths,
    const PathSegment *pathSegments,
    const Geom *geoms,
    int geoms_size,
    const Mesh *meshes,
    int *indices,
    const glm::vec3 *points,
    const glm::vec2 *uvs,
    ShadeableIntersection *intersections)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (path_index >= num_paths)
    {
        return;
    }

    intersections[path_index] = queryIntersection(pathSegments[path_index].ray, geoms, geoms_size, meshes, indices, points, uvs);
}

__global__ void chooseLights(
    int numLights,
    int numPaths,
    int iter,
    int *lightIndices)
{
    size_t idx = blockIdx.x * blockDim.x * threadIdx.x;
    if (idx >= numPaths)
    {
        return;
    }
    auto rng = makeSeededRandomEngine(iter, idx, 0);
    thrust::uniform_int_distribution<int> dist(0, numLights - 1);
    lightIndices[idx] = dist(rng);
}

// __global__ void shadeMaterialDirect(
//     int num_paths,
//     int iter,
//     int lightCount,
//     ShadeableIntersection *shadeableIntersections,
//     Geom *geoms,
//     int geomsSize,
//     Mesh *meshes,
//     glm::vec3 *points,
//     int *indices,
//     PathSegment *pathSegments,
//     Material *materials)
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx >= num_paths)
//     {
//         return;
//     }

//     PathSegment &segment = pathSegments[idx];

//     ShadeableIntersection intersection = shadeableIntersections[idx];
//     if (intersection.t <= 0)
//     {
//         segment.remainingBounces = 0;
//         return;
//     }

//     const Material &material = materials[intersection.materialId];
//     if (glm::length(material.emittance.value) > 0.f)
//     {
//         segment.radiance += segment.throughput * material.emittance.value;
//         segment.remainingBounces = 0;
//     }

//     glm::vec3 viewPoint = getPointOnRay(segment.ray, intersection.t);

//     auto rng = makeSeededRandomEngine(iter, idx, 0);
//     thrust::uniform_int_distribution<int> dist(0, lightCount - 1);
//     int lightIndex = dist(rng);

//     Sample lightSample = sampleLight(viewPoint, geoms[lightIndex], materials, rng);
//     Ray checkRay{.origin = viewPoint + EPSILON * lightSample.incomingDirection, .direction = lightSample.incomingDirection};
//     // int shadowResult = queryIntersectionGeometryIndex(checkRay, geoms, geomsSize, tris, trisSize);
//     // TODO: bring back shadow casting once BVHs are done
//     int shadowResult = lightIndex;
//     if (shadowResult != lightIndex)
//     {
//         lightSample.value = glm::vec3(0, 0, 0);
//     }
//     segment.throughput = getBsdf(materials[intersection.materialId], intersection.surfaceNormal, lightSample.incomingDirection, segment.ray.direction);
//     segment.radiance = segment.throughput * lightSample.value / lightSample.pdf;
// }

// The actual entrypoint for shading a material
__global__ void shadeMaterialSimple(
    int iter,
    int num_paths,
    ShadeableIntersection *shadeableIntersections,
    Geom *geoms,
    PathSegment *pathSegments,
    Material *materials)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths)
    {
        return;
    }

    PathSegment &segment = pathSegments[idx];

#ifndef STREAMCOMPACT
    if (segment.remainingBounces <= 0)
    {
        return;
    }
#endif

    const ShadeableIntersection &intersection = shadeableIntersections[idx];
    const Material &material = materials[intersection.materialId];
    if (material.emissiveStrength > 0.f) {
        glm::vec3 emittance = sampleTexture(material.emittance, intersection.emissiveUv) * material.emissiveStrength;
        segment.radiance += segment.throughput * emittance;
    }

    glm::vec3 intersect = getPointOnRay(segment.ray, intersection.t);
    thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, segment.remainingBounces);
    scatterRay(segment, intersect, intersection.surfaceNormal, material, intersection.albedoUv, rng);

    if (glm::length(segment.throughput) < EPSILON) {
        segment.remainingBounces = 0;
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 *image, PathSegment *iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.radiance;
    }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment *dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    int active_paths = num_paths;

    // if (hst_scene->useDirectLighting)
    // {
    //     hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
    //     dim3 numblocksPathSegmentTracing = (active_paths + blockSize1d - 1) / blockSize1d;
    //     computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(
    //         depth,
    //         active_paths,
    //         dev_paths,
    //         dev_geoms,
    //         hst_scene->geoms.size(),
    //         dev_meshes,
    //         dev_indices,
    //         dev_points,
    //         dev_uvs,
    //         dev_intersections);
    //     hipDeviceSynchronize();
    //     depth++;
    //     shadeMaterialDirect<<<numblocksPathSegmentTracing, blockSize1d>>>(
    //         num_paths,
    //         iter,
    //         hst_scene->numLights,
    //         dev_intersections,
    //         dev_geoms,
    //         hst_scene->geoms.size(),
    //         dev_meshes,
    //         dev_points,
    //         dev_indices,
    //         dev_paths,
    //         dev_materials);

    //     if (guiData != NULL)
    //     {
    //         guiData->TracedDepth = depth;
    //     }
    // }
    // else
    // {
        while (!iterationComplete)
        {
            // clean shading chunks
            hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

            // tracing
            dim3 numblocksPathSegmentTracing = (active_paths + blockSize1d - 1) / blockSize1d;
            computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(
                depth,
                active_paths,
                dev_paths,
                dev_geoms,
                hst_scene->geoms.size(),
                dev_meshes,
                dev_indices,
                dev_points,
                dev_uvs,
                dev_intersections);
            checkCUDAError("trace one bounce");
            hipDeviceSynchronize(); //TODO: remove this sync?
            
            // Sort by material
            thrust::sort_by_key(thrust::device,
                thrust::device_pointer_cast(dev_intersections),
                thrust::device_pointer_cast(dev_intersections) + active_paths,
                thrust::device_pointer_cast(dev_paths),
                CmpMaterial()           
            );

            // Terminate any paths that did not hit a material
            active_paths = thrust::partition_point(thrust::device, dev_intersections, dev_intersections + active_paths, IntersectionValid()) - dev_intersections;
            depth++;

            hipDeviceSynchronize();
            shadeMaterialSimple<<<numblocksPathSegmentTracing, blockSize1d>>>(
                iter,
                active_paths,
                dev_intersections,
                dev_geoms,
                dev_paths,
                dev_materials);

#ifdef STREAMCOMPACT
            active_paths = thrust::partition(thrust::device, dev_paths, dev_paths + active_paths, PathActive()) - dev_paths;
            iterationComplete = active_paths == 0 || depth > traceDepth;
#endif
#ifndef STREAMCOMPACT
            iterationComplete = depth > traceDepth;
#endif

            if (guiData != NULL)
            {
                guiData->TracedDepth = depth;
            }
        }
    // }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
               pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}