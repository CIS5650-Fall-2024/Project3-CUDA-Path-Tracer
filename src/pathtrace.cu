#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
#include <>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

static Scene* hst_scene = NULL;
static glm::vec3* dev_image = NULL;
static GuiDataContainer* guiData = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
static glm::vec3* dev_vertices = NULL;
static Mesh* dev_meshes = NULL;
static glm::vec3* dev_normals = NULL;
static glm::vec2* dev_texcoords = NULL;

#if BVH
static BVHNode* dev_bvh = NULL;
#endif

#if OIDN
#define EMA_ALPHA 0.2f
#define DENOISE_INTERVAL 100

#include <OpenImageDenoise/oidn.hpp>

static glm::vec3* dev_denoised = NULL;
static glm::vec3* dev_albedo = NULL;
static glm::vec3* dev_normal = NULL;

void denoise()
{
    int width = hst_scene->state.camera.resolution.x,
        height = hst_scene->state.camera.resolution.y;

    // Create an Intel Open Image Denoise device
    oidn::DeviceRef device = oidn::newDevice();
    device.commit();

    // Create a filter for denoising a beauty (color) image using prefiltered auxiliary images too
    oidn::FilterRef filter = device.newFilter("RT"); // generic ray tracing filter
    filter.setImage("color", dev_image, oidn::Format::Float3, width, height); // beauty
    filter.setImage("albedo", dev_albedo, oidn::Format::Float3, width, height); // auxiliary
    filter.setImage("normal", dev_normal, oidn::Format::Float3, width, height); // auxiliary
    filter.setImage("output", dev_denoised, oidn::Format::Float3, width, height); // denoised beauty
    filter.set("hdr", true); // image is HDR
    filter.set("cleanAux", true); // auxiliary images will be prefiltered
    filter.commit();

    // Create a separate filter for denoising an auxiliary albedo image (in-place)
    oidn::FilterRef albedoFilter = device.newFilter("RT"); // same filter type as for beauty
    albedoFilter.setImage("albedo", dev_albedo, oidn::Format::Float3, width, height);
    albedoFilter.setImage("output", dev_albedo, oidn::Format::Float3, width, height);
    albedoFilter.commit();

    // Create a separate filter for denoising an auxiliary normal image (in-place)
    oidn::FilterRef normalFilter = device.newFilter("RT"); // same filter type as for beauty
    normalFilter.setImage("normal", dev_normal, oidn::Format::Float3, width, height);
    normalFilter.setImage("output", dev_normal, oidn::Format::Float3, width, height);
    normalFilter.commit();

    // Prefilter the auxiliary images
    albedoFilter.execute();
    normalFilter.execute();

    // Filter the beauty image
    filter.execute();

    // Check for errors
    const char* errorMessage;
    if (device.getError(errorMessage) != oidn::Error::None)
        std::cout << "Error: " << errorMessage << std::endl;
}

__global__
void copyFirstTraceResult(
    PathSegment* pathSegments, int num_paths,
    ShadeableIntersection* shadeableIntersections,
    glm::vec3* albedo, glm::vec3* normal)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths) {
        PathSegment pathSegment = pathSegments[idx];
        ShadeableIntersection intersection = shadeableIntersections[idx];

        albedo[pathSegment.pixelIndex] = pathSegment.color;
        normal[pathSegment.pixelIndex] = intersection.surfaceNormal;
    }
}

__global__
void emaMergeDenoisedAndImage(int pixelcount, glm::vec3* image, glm::vec3* denoised)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < pixelcount) {
        // exponential moving average
        image[idx] = image[idx] * (1 - EMA_ALPHA) + denoised[idx] * EMA_ALPHA;
    }
}
#endif

void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

struct isRayAlive
{
    __host__ __device__ bool operator()(const PathSegment& path)
    {
        return path.remainingBounces > 0;
    }
};

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    //Mesh data
    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_vertices, scene->vertices.size() * sizeof(glm::vec3));
    hipMemcpy(dev_vertices, scene->vertices.data(), scene->vertices.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);

    hipMalloc(&dev_meshes, scene->meshes.size() * sizeof(Mesh));
    hipMemcpy(dev_meshes, scene->meshes.data(), scene->meshes.size() * sizeof(Mesh), hipMemcpyHostToDevice);

    hipMalloc(&dev_normals, scene->normals.size() * sizeof(glm::vec3));
    hipMemcpy(dev_normals, scene->normals.data(), scene->normals.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);

    hipMalloc(&dev_texcoords, scene->texcoords.size() * sizeof(glm::vec2));
    hipMemcpy(dev_texcoords, scene->texcoords.data(), scene->texcoords.size() * sizeof(glm::vec2), hipMemcpyHostToDevice);

#if BVH
    hipMalloc(&dev_bvh, scene->bvh.size() * sizeof(BVHNode));
    hipMemcpy(dev_bvh, scene->bvh.data(), scene->bvh.size() * sizeof(BVHNode), hipMemcpyHostToDevice);
#endif

#if OIDN
    hipMalloc(&dev_denoised, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_denoised, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_albedo, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_albedo, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_normal, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_normal, 0, pixelcount * sizeof(glm::vec3));
#endif

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image); 
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    hipFree(dev_texcoords);
    hipFree(dev_normals);
    hipFree(dev_vertices);
    hipFree(dev_meshes);

#if BVH
    hipFree(dev_bvh);
#endif

#if OIDN
    hipFree(dev_denoised);
    hipFree(dev_albedo);
    hipFree(dev_normal);
#endif


    checkCUDAError("pathtraceFree");
}

__device__ glm::vec3 checkerboard(glm::vec2 uv)
{
    if ((int)(uv.x * 10) % 2 == (int)(uv.y * 10) % 2)
        return glm::vec3(.2f);
    else
        return glm::vec3(.8f);
}
__device__ glm::vec3 palettes(glm::vec2 uv)
{
    glm::vec3 a(0.5, 0.5, 0.5), b(0.5, 0.5, 0.5), c(1.0, 1.0, 1.0), d(0.00, 0.33, 0.67);
    return a + b * glm::cos(TWO_PI * (c * glm::length(uv) + d));
}

__host__ __device__
glm::vec2 ConcentricSampleDisk(const glm::vec2& u)
{
    glm::vec2 uOffset = 2.0f * u - glm::vec2(1.0f, 1.0f);

    if (uOffset.x == 0.0f && uOffset.y == 0.0f)
    {
        return glm::vec2(0.0f, 0.0f);
    }

    float theta, r;
    if (glm::abs(uOffset.x) > glm::abs(uOffset.y))
    {
        r = uOffset.x;
        theta = PI_OVER_FOUR * (uOffset.y / uOffset.x);
    }
    else
    {
        r = uOffset.y;
        theta = PI_OVER_TWO - PI_OVER_FOUR * (uOffset.x / uOffset.y);
    }
    return r * glm::vec2(glm::cos(theta), glm::sin(theta));
}


/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        //stochastic sampled antialiasing (SSAA)
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
        thrust::uniform_real_distribution<float> u01(0, 1);
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x + u01(rng) - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y + u01(rng) - (float)cam.resolution.y * 0.5f)
        );

        // Depth of field automatically Enabled for camera with LENSRADIUS and FOCALDIS
        if (cam.lensRadius > 0)
        {
            glm::vec2 pLens = cam.lensRadius * ConcentricSampleDisk(glm::vec2(u01(rng), u01(rng)));
            float ft = cam.focalDistance / glm::dot(cam.view, segment.ray.direction);
            glm::vec3 pFocus = segment.ray.origin + segment.ray.direction * ft;
            segment.ray.origin += cam.right * pLens.x + cam.up * pLens.y;
            segment.ray.direction = glm::normalize(pFocus - segment.ray.origin);
        }

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    ShadeableIntersection* intersections,
#if BVH
    BVHNode* bvh,
#endif
    Mesh* meshes
    , glm::vec3* vertices
    , glm::vec3* normals
    , glm::vec2* texcoords)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        glm::vec2 uv;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;
        glm::vec2 tmp_uv;

        int tmp_material_index;
        glm::vec2 tmp_texcoord;

        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside, tmp_uv);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside,tmp_uv);
            }
            else if (geom.type == MESH) 
            {
#if BVH
                t = meshIntersectionTestBVH(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside, tmp_uv,
                    bvh, meshes, vertices, normals, texcoords, tmp_material_index);

#else
                t = meshIntersectionTestNaive(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside, tmp_uv, 
                    meshes, vertices, normals, texcoords, tmp_material_index);
#endif        
            }
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
                uv = tmp_uv;
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            // The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
            intersections[path_index].uv = uv;
        }
    }
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeMaterial(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        if (intersection.t > 0.0f)
        {
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;
            if (material.emittance > 0.0f) {
                pathSegments[idx].color *= (materialColor * material.emittance);
                pathSegments[idx].remainingBounces = 0;
            }
            else {
                switch (material.procedualTextureID) {
                     case 1: material.color = checkerboard(intersection.uv); break;
                     case 2: material.color = palettes(intersection.uv); break;
                     default: break;
                }
                float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
                glm::vec3 intersect = intersection.t * pathSegments[idx].ray.direction + pathSegments[idx].ray.origin;
                scatterRay(pathSegments[idx], intersect, intersection.surfaceNormal, material, rng);
            }
        }
        else {
            pathSegments[idx].color = glm::vec3(0.0f);
            pathSegments[idx].remainingBounces = 0;
        }
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    const int blockSize1d = 128;

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_intersections
#if BVH
            , dev_bvh
#endif 
            , dev_meshes, dev_vertices, dev_normals, dev_texcoords
        );
        hipDeviceSynchronize();
        depth++;

        // --- Shading Stage ---
#ifndef SORT_MATERIAL_ID
        thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, materialsCmp());
#endif
        shadeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials
        );

#if OIDN
        if (depth == 1 && (iter % DENOISE_INTERVAL == 0 || iter == hst_scene->state.iterations))
            copyFirstTraceResult << <numblocksPathSegmentTracing, blockSize1d >> > (
                dev_paths, num_paths, dev_intersections, dev_albedo, dev_normal);
#endif

#ifdef STREAM_COMPACTION
        num_paths = thrust::partition(thrust::device,
            dev_paths, dev_paths + num_paths, isRayAlive()) - dev_paths;
#endif

        iterationComplete = depth == traceDepth || num_paths == 0;

        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

#ifdef STREAM_COMPACTION
    num_paths = dev_path_end - dev_paths;
#endif

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

#if OIDN
    if (iter % DENOISE_INTERVAL == 0 && iter != 0)
    {
        denoise();
        emaMergeDenoisedAndImage << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_image, dev_denoised);
    }
    else if (iter == hst_scene->state.iterations)
    {
        denoise();
        std::swap(dev_image, dev_denoised);
    }
#endif

    // --- Rendering Stage ---

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
