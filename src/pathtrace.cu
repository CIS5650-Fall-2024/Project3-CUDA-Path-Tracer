#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include <>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h" 

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        // TODO: implement antialiasing by jittering the ray
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
        );

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
        segment.isFinished = false; 
    }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    ShadeableIntersection* intersections)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;

        // naive parse through global geoms

        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            // TODO: add more intersection tests here... triangle? metaball? CSG?

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            // The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
        }
    }
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        if (intersection.t > 0.0f) // if the intersection exists...
        {
          // Set up the RNG
          // LOOK: this is how you use thrust's RNG! Please look at
          // makeSeededRandomEngine as well.
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;

            // If the material indicates that the object was a light, "light" the ray
            if (material.emittance > 0.0f) {
                pathSegments[idx].color *= (materialColor * material.emittance);
            }
            // Otherwise, do some pseudo-lighting computation. This is actually more
            // like what you would expect from shading in a rasterizer like OpenGL.
            // TODO: replace this! you should be able to start with basically a one-liner
            else {
                float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
                pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
                pathSegments[idx].color *= u01(rng); // apply some noise because why not
            }
            // If there was no intersection, color the ray black.
            // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
            // used for opacity, in which case they can indicate "no opacity".
            // This can be useful for post-processing and image compositing.
        }
        else {
            pathSegments[idx].color = glm::vec3(0.0f);
        }
    }
}

__device__ glm::vec3 squareToHemisphereCosine(glm::vec2 xi) {
  float x = xi.x;
  float y = xi.y;
  if (x == 0 && y == 0)
    return glm::vec3(0, 0, 0);

  float phi = 0.f;
  float radius = 1.f;
  float a = (2.f * x) - 1.f;
  float b = (2.f * y) - 1.f;

  // Uses squares instead of absolute values
  if ((a * a) > (b * b)) {
    // Top half
    radius *= a;
    phi = (PI / 4) * (b / a);
  }
  else {
    // Bottom half
    radius *= b;
    phi = (PI / 2) - ((PI / 4) * (a / b));
  }

  // Map the distorted Polar coordinates (phi,radius)
  // into the Cartesian (x,y) space
  glm::vec3 disc(0.f, 0.f, 0.f);
  disc.x = glm::cos(phi) * radius;
  disc.y = glm::sin(phi) * radius;

  // I think this ensures this is a hemisphere and not a sphere ? 
  disc.z = glm::sqrt(1.f - (disc.x * disc.x) - (disc.y * disc.y));

  return disc;
}

__device__ void localToWorld(const glm::vec3& normal, glm::vec3& vec) {
  glm::vec3 tangent, bitangent;

  // create coordinate system from normal 
  if (glm::abs(normal.x) > glm::abs(normal.y)) {
    tangent = glm::vec3(-normal.z, 0, normal.x) / glm::sqrt(normal.x * normal.x + normal.z * normal.z);
  }
  else {
    tangent = glm::vec3(0, normal.z, -normal.y) / glm::sqrt(normal.y * normal.y + normal.z * normal.z);
  }
    
  bitangent = glm::cross(normal, tangent);

  vec = glm::mat3(tangent, bitangent, normal) * vec; 
}

__device__ void worldToLocal(const glm::vec3& normal, glm::vec3& vec) {
  glm::vec3 tangent, bitangent;

  // create coordinate system from normal 
  if (glm::abs(normal.x) > glm::abs(normal.y)) {
    tangent = glm::vec3(-normal.z, 0, normal.x) / glm::sqrt(normal.x * normal.x + normal.z * normal.z);
  }
  else {
    tangent = glm::vec3(0, normal.z, -normal.y) / glm::sqrt(normal.y * normal.y + normal.z * normal.z);
  }

  bitangent = glm::cross(normal, tangent);

  vec = glm::transpose(glm::mat3(tangent, bitangent, normal)) * vec;
}


// TODO: if pathsegment has no more remaining bounces, do nothing
__global__ void shadeMaterial(
  int iter,
  int num_paths,
  ShadeableIntersection* shadeableIntersections,
  PathSegment* pathSegments,
  Material* materials)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= num_paths) {
    return; 
  }

  PathSegment pathSegment = pathSegments[idx];

  if (pathSegment.isFinished) {
    return; 
  }

  ShadeableIntersection intersection = shadeableIntersections[idx];

  if (intersection.t <= 0.0f) {
    pathSegment.color = glm::vec3(0.0f);
    pathSegment.isFinished = true; 
  } 
  else if (pathSegment.remainingBounces <= 0) {
    pathSegment.color = glm::vec3(0.0f);
    pathSegment.isFinished = true; 
  }
  else {
    Material material = materials[intersection.materialId];
    float pdf = 0.f;
    glm::vec2 xi; 

    // compute a random vec2
    {
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegment.remainingBounces);
      thrust::uniform_real_distribution<float> u01(0, 1);
      xi.x = u01(rng); 
      xi.y = u01(rng); 
    }

    if (material.emittance > 0.0f) {                      // LIGHT
      // TODO: emmitance * current throughput (?)
      pathSegment.color *= (material.color * material.emittance);
      // pathSegment.color /= (float)iter; 
      pathSegment.isFinished = true; 
    }
#if 0 
    else if (material.hasReflective > 0.f) {              // SPECULAR
      // TODO: implement perfect specular
      pathSegment.color = glm::vec3(0., 0., 0.);
      pathSegment.remainingBounces = 0;
    }
#endif
    else {                                                // DIFFUSE
      // generate random direction in hemisphere
      glm::vec3 wi = squareToHemisphereCosine(xi);

      glm::vec3 local_wo = -pathSegment.ray.direction; 
      worldToLocal(intersection.surfaceNormal, -pathSegment.ray.direction); 

      // From https://pbr-book.org/4ed/Reflection_Models/Diffuse_Reflection, 
      // to be honest I'm not sure what this does yet.
      // I guess, if we're intersecting with a point that's 
      // "upside down", we have to flip our generated iw too?
      if (local_wo.z < 0) wi.z *= -1;

      // get the pdf (square to hemisphere cosine)
      pdf = glm::abs(wi.z) * INV_PI;

      if (glm::isnan(pdf) || pdf < EPSILON) {
        pathSegment.isFinished = true;
        pathSegment.color = glm::vec3(0.);
        pathSegments[idx] = pathSegment;
        return;
      }

      // convert vec3 into the world coordinate system (using surface normal)
      localToWorld(intersection.surfaceNormal, wi);
      wi = glm::normalize(wi); 

      // get the diffuse color (albedo * INV_PI)
      glm::vec3 bsdfValue = material.color * INV_PI;

      // update throughput
      pathSegment.color *= bsdfValue * glm::abs(glm::dot(intersection.surfaceNormal, wi)) / pdf;

      // new ray for the next bounce
      pathSegment.ray.origin = pathSegment.ray.origin + (intersection.t * pathSegment.ray.direction); 
      pathSegment.ray.origin += EPSILON * wi;   // slightly offset the ray origin in the direction of the ray direction
      pathSegment.ray.direction = wi; 

      --pathSegment.remainingBounces;
    }
  }

  // read back into global memory
  pathSegments[idx] = pathSegment;
}


// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    //const int traceDepth = 1; // DEBUG
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    while (depth < traceDepth)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_intersections
        );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

        // TODO:
        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        // path segments that have been reshuffled to be contiguous in memory.

        shadeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials
        );

        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
