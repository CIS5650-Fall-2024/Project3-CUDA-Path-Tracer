#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

#include <thrust/device_vector.h>
#include <thrust/logical.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)


#define USE_RUSSIAN_ROULETTE 1

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;

static Material* dev_materials = NULL;
static Geom* dev_geoms = NULL;
static Geom* dev_lights = NULL;
static Triangle* dev_geomTriangles = NULL;
static Triangle* dev_lightTriangles = NULL;
static int* dev_totalNumberOfLights = NULL;

static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;

struct sortMaterialCondition
{
    __host__ __device__
        bool operator()(const ShadeableIntersection& s1, const ShadeableIntersection& s2)
    {
        return s1.materialId < s2.materialId;
    }
};

struct has_remaining_bounces
{
    __host__ __device__
        bool operator()(const PathSegment& path)
    {
        return path.remainingBounces > 0;
    }
};

void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void initialiseTriangles(Triangle* dev_triangles, std::vector<Geom>& geometries, const int totalNumberOfGeom)
{   
    if (totalNumberOfGeom == 0) {
        return;
    }

    int totalNumberOfTriangles = 0;
    for (int i = 0; i < totalNumberOfGeom; i++) {
        if (geometries[i].type == MESH) {
            totalNumberOfTriangles += geometries[i].numTriangles;
        }
    }

    if (totalNumberOfTriangles == 0) {
        return;
    }

    hipMalloc(&dev_triangles, totalNumberOfTriangles * sizeof(Triangle));
    int offset = 0;
    for (int i = 0; i < totalNumberOfGeom; i++) {
        if (geometries[i].type == MESH) {
            // Copy each geometry's triangles to the device memory
            hipMemcpy(dev_triangles + offset, geometries[i].triangles, geometries[i].numTriangles * sizeof(Triangle), hipMemcpyHostToDevice);
            
            // Update the device pointer in the geometry struct to point to device memory
            geometries[i].devTriangles = dev_triangles + offset;
            
            // Move the offset by the number of triangles in this geometry
            offset += geometries[i].numTriangles;
        }
    }
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    int totalNumberOfGeom = scene->geoms.size();
    initialiseTriangles(dev_geomTriangles, scene->geoms, totalNumberOfGeom); // Must appear before initializing dev_geoms
    hipMalloc(&dev_geoms, totalNumberOfGeom * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    int totalNumberOfLights = scene->lights.size();
    initialiseTriangles(dev_lightTriangles, scene->lights, totalNumberOfLights); // Must appear before initializing dev_lights
    hipMalloc(&dev_lights, totalNumberOfLights * sizeof(Geom));
    hipMemcpy(dev_lights, scene->lights.data(), scene->lights.size() * sizeof(Geom), hipMemcpyHostToDevice);
    hipMalloc(&dev_totalNumberOfLights, sizeof(int));
    hipMemcpy(dev_totalNumberOfLights, &totalNumberOfLights, sizeof(int), hipMemcpyHostToDevice);

    // We've already got the triangles in the device memory, so we can delete them from the host memory
    delete scene->geoms.data()->triangles;

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_lights);
    hipFree(dev_geomTriangles);
    hipFree(dev_lightTriangles);
    hipFree(dev_totalNumberOfLights);
    hipFree(dev_materials);
    hipFree(dev_intersections);

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x >= cam.resolution.x || y >= cam.resolution.y) {
        return;
    }

    int index = x + (y * cam.resolution.x);

    thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, traceDepth);
    thrust::uniform_real_distribution<float> u01(0, 1);

    PathSegment& segment = pathSegments[index];

    segment.ray.origin = cam.position;
    segment.color = glm::vec3(1.0f);

    glm::vec2 offset = glm::vec2(0.5f * (u01(rng) * 2.0f - 1.0f), 0.5f * (u01(rng) * 2.0f - 1.0f));
    segment.ray.direction = glm::normalize(cam.view
        - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f + offset[0])
        - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f + offset[1])
    );

    segment.pixelIndex = index;
    segment.remainingBounces = traceDepth;
    segment.hasHitLight = false;
    segment.eta = 1.0f;
}

// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    ShadeableIntersection* intersections)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;

        // naive parse through global geoms
        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == MESH) {
                t = meshIntersectionTestNaive(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            // The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
        }
    }
}

__global__ void shadeNaive(
    int iter,
    int depth,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials) {
    // As long as we enter here, it means the ray has remaining bounces > 0
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) {
        return;
    }

    ShadeableIntersection intersection = shadeableIntersections[idx];
    PathSegment& pathSegment = pathSegments[idx];
    if (intersection.t <= 0.0f) {
        pathSegment.color = glm::vec3(0.0f);
        pathSegment.remainingBounces = 0;
        return;
    }

    Material material = materials[intersection.materialId];
    glm::vec3 materialColor = material.color;
    
    if (material.emittance > 0.0f) {
        pathSegment.color *= materialColor * material.emittance;
        pathSegment.remainingBounces = 0;
        pathSegment.hasHitLight = true;
    }
    else {
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, depth);
        thrust::uniform_real_distribution<float> u01(0, 1);

        glm::vec3 oldIntersect = getPointOnRay(pathSegment.ray, intersection.t);
        glm::vec3 surfaceNormal = glm::normalize(intersection.surfaceNormal);
        glm::vec3 woW = -pathSegment.ray.direction;
        glm::vec3 wiW;
        glm::vec3 c;
        float pdf;
        float eta;
        
        scatterRay(pathSegment, woW, surfaceNormal, wiW, pdf, c, eta, material, rng); 

        pathSegment.ray.direction = wiW; // wiW should already be normalized
        // Without the offset, when the ray immediately intersects the surface it originated from, the refraction calculations may fail or yield invalid results, such as:
        // Total Internal Reflection: The refracted ray might get treated as a reflective ray due to intersection problems, resulting in no transmitted light.
        // Black Pixels: The lack of refraction or valid light contribution can result in areas appearing black, as seen in your case.
        pathSegment.ray.origin = oldIntersect + pathSegment.ray.direction * 0.01f;
        pathSegment.color *= c; 
        
        #if (USE_RUSSIAN_ROULETTE) // Possibly terminate the path with Russian roulette
            if (depth > 3) {
                // So that the ray can bounce for a bit before we start terminating it
                float maxComponent = fmaxf(c.x, fmaxf(c.y, c.z));
                float survivalProbability = u01(rng);
                float eta_sq = eta * eta;
                float q = fminf(maxComponent * eta_sq, 0.99f);
                
                if (q < survivalProbability) {
                    pathSegment.remainingBounces = 0;
                    return;
                }
                else {
                    pathSegment.color /= q;
                }
            }        
        #endif
        
        pathSegment.remainingBounces--;
    }
}

__host__ __device__ int sampleTriangleFromMesh(const Triangle* triangles, const int numTriangles, const float randVal) {
    // Perform a binary search over the CDF to find the corresponding triangle
    int left = 0;
    int right = numTriangles - 1;

    while (left < right) {
        int mid = left + (right - left) / 2;
        if (randVal < triangles[mid].cdf) {
            right = mid;
        }
        else {
            left = mid + 1;
        }
    }

    return left;
}

// Sample a light source and return the sampled point in world space
__host__ __device__ glm::vec3 sampleLight(const int totalNumberOfLights, const Geom* lights, const Material* mats, thrust::default_random_engine &rng, glm::vec3 &sampledPointWorld, glm::vec3 &sampledNormalWorld, float &pdf) {
    thrust::uniform_real_distribution<float> u01(0, 1);
    // Randomly sample an emitter from the list, ensuring the index doesn't exceed totalLights - 1
    int light_idx = min(int(u01(rng) * totalNumberOfLights), totalNumberOfLights - 1);
    // Get the emitter from the list
    Geom light = lights[light_idx];

    // So far we assum only a mesh can be a light and it's an area light
    if (light.type == MESH) {
        glm::vec3 samples = glm::vec3(u01(rng), u01(rng), u01(rng));
        int triangleIdx = sampleTriangleFromMesh(light.devTriangles, light.numTriangles, samples.x);
        Triangle lightTriangle = light.devTriangles[triangleIdx];
        float alpha = 1.0f - sqrt(1.0f - samples.y);
        float beta = samples.z * sqrt(1.0f - samples.y);
        float gamma = 1.0f - alpha - beta;

        glm::vec3 sampledPointLocal = alpha * lightTriangle.points[0] + beta * lightTriangle.points[1] + gamma * lightTriangle.points[2];
        glm::vec3 sampledNormalLocal = glm::normalize(alpha * lightTriangle.normals[0] + beta * lightTriangle.normals[1] + gamma * lightTriangle.normals[2]);

        sampledPointWorld = multiplyMV(light.transform, glm::vec4(sampledPointLocal, 1.0f));
        sampledNormalWorld = glm::normalize(multiplyMV(light.invTranspose, glm::vec4(sampledNormalLocal, 0.0f)));
        pdf = 1.0f / light.area / totalNumberOfLights;
        Material lightMaterial = mats[light.materialid];
        return lightMaterial.color * lightMaterial.emittance;
    }

    return glm::vec3(0.0f);
}

__host__ __device__ bool isRayOccluded(const int geomsSize, const Geom* geoms, Ray &ray) {
    float t;
    bool outside = true;
    glm::vec3 tmp_intersect;
    glm::vec3 tmp_normal;

    for (int i = 0; i < geomsSize; i++) {
        Geom geom = geoms[i];

        if (geom.type == CUBE) {
            t = boxIntersectionTest(geom, ray, tmp_intersect, tmp_normal, outside);
        }
        else if (geom.type == SPHERE) {
            t = sphereIntersectionTest(geom, ray, tmp_intersect, tmp_normal, outside);
        }
        else if (geom.type == MESH) {
            t = meshIntersectionTestNaive(geom, ray, tmp_intersect, tmp_normal, outside);
        }

        if (t > 0.0f) {
            return true;
        }
    }

    return false;
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        if (iterationPath.hasHitLight) {
            image[iterationPath.pixelIndex] += iterationPath.color;  
        }   
    }
}

__global__ void computeIsIntersected(int num_paths, int* isIntersected, const ShadeableIntersection* intersections)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) {
        return;
    }

    isIntersected[idx] = intersections[idx].t != -1.0f;
}

void partitionRays(int &num_paths, PathSegment* dev_paths, const ShadeableIntersection* dev_intersections) {
    thrust::device_ptr<PathSegment> dev_ptr(dev_paths);
    thrust::device_ptr<PathSegment> dev_ptr_end = thrust::stable_partition(thrust::device, dev_ptr, dev_ptr + num_paths, has_remaining_bounces());
    hipDeviceSynchronize();
    num_paths = dev_ptr_end - dev_ptr;
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int total_num_paths = dev_path_end - dev_paths;
    int num_paths = total_num_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks
    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_intersections
        );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

        // Sort materials by type
        thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, sortMaterialCondition());
        hipDeviceSynchronize();

        shadeNaive<<<numblocksPathSegmentTracing, blockSize1d>>>(
            iter,
            depth,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials
        );
        hipDeviceSynchronize();

        // compact paths
        partitionRays(num_paths, dev_paths, dev_intersections);

        iterationComplete = (depth >= traceDepth) || (num_paths == 0);
        
        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(total_num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}