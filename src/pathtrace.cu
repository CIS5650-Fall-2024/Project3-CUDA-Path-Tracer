#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
#include <>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

static Scene* hst_scene = NULL;
static glm::vec3* dev_image = NULL;
static GuiDataContainer* guiData = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
#if BVH
static BVHNode* dev_bvh = NULL;
#endif
static glm::vec3* dev_vertices = NULL;
static Mesh* dev_meshes = NULL;
static glm::vec3* dev_normals = NULL;
static glm::vec2* dev_texcoords = NULL;

void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

struct isRayAlive
{
    __host__ __device__ bool operator()(const PathSegment& path)
    {
        return path.remainingBounces > 0;
    }
};

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    //Mesh data
    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_vertices, scene->vertices.size() * sizeof(glm::vec3));
    hipMemcpy(dev_vertices, scene->vertices.data(), scene->vertices.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);

    hipMalloc(&dev_meshes, scene->meshes.size() * sizeof(Mesh));
    hipMemcpy(dev_meshes, scene->meshes.data(), scene->meshes.size() * sizeof(Mesh), hipMemcpyHostToDevice);

#if BVH
    hipMalloc(&dev_bvh, scene->bvh.size() * sizeof(BVHNode));
    hipMemcpy(dev_bvh, scene->bvh.data(), scene->bvh.size() * sizeof(BVHNode), hipMemcpyHostToDevice);
#endif

    hipMalloc(&dev_normals, scene->normals.size() * sizeof(glm::vec3));
    hipMemcpy(dev_normals, scene->normals.data(), scene->normals.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);

    hipMalloc(&dev_texcoords, scene->texcoords.size() * sizeof(glm::vec2));
    hipMemcpy(dev_texcoords, scene->texcoords.data(), scene->texcoords.size() * sizeof(glm::vec2), hipMemcpyHostToDevice);



    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created
    hipFree(dev_texcoords);
    hipFree(dev_normals);
    hipFree(dev_bvh);
    hipFree(dev_vertices);
    hipFree(dev_meshes);

    checkCUDAError("pathtraceFree");
}

__device__ glm::vec3 checkerboard(glm::vec2 uv)
{
    if ((int)(uv.x * 10) % 2 == (int)(uv.y * 10) % 2)
        return glm::vec3(.2f);
    else
        return glm::vec3(.8f);
}
__device__ glm::vec3 palettes(glm::vec2 uv)
{
    glm::vec3 a(0.5, 0.5, 0.5), b(0.5, 0.5, 0.5), c(1.0, 1.0, 1.0), d(0.00, 0.33, 0.67);
    return a + b * glm::cos(TWO_PI * (c * glm::length(uv) + d));
}

__host__ __device__
glm::vec2 ConcentricSampleDisk(const glm::vec2& u)
{
    glm::vec2 uOffset = 2.0f * u - glm::vec2(1.0f, 1.0f);

    if (uOffset.x == 0.0f && uOffset.y == 0.0f)
    {
        return glm::vec2(0.0f, 0.0f);
    }

    float theta, r;
    if (glm::abs(uOffset.x) > glm::abs(uOffset.y))
    {
        r = uOffset.x;
        theta = PI_OVER_FOUR * (uOffset.y / uOffset.x);
    }
    else
    {
        r = uOffset.y;
        theta = PI_OVER_TWO - PI_OVER_FOUR * (uOffset.x / uOffset.y);
    }
    return r * glm::vec2(glm::cos(theta), glm::sin(theta));
}


/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        //stochastic sampled antialiasing (SSAA)
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
        thrust::uniform_real_distribution<float> u01(0, 1);
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x + u01(rng) - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y + u01(rng) - (float)cam.resolution.y * 0.5f)
        );

        // Depth of field automatically Enabled for camera with LENSRADIUS and FOCALDIS
        if (cam.lensRadius > 0)
        {
            glm::vec2 pLens = cam.lensRadius * ConcentricSampleDisk(glm::vec2(u01(rng), u01(rng)));
            float ft = cam.focalDistance / glm::dot(cam.view, segment.ray.direction);
            glm::vec3 pFocus = segment.ray.origin + segment.ray.direction * ft;
            segment.ray.origin += cam.right * pLens.x + cam.up * pLens.y;
            segment.ray.direction = glm::normalize(pFocus - segment.ray.origin);
        }

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    ShadeableIntersection* intersections,
#if BVH
    BVHNode* bvh
#endif
    , Mesh* meshes
    , glm::vec3* vertices
    , glm::vec3* normals
    , glm::vec2* texcoords)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        glm::vec2 uv;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;
        glm::vec2 tmp_uv;

        int tmp_material_index;
        glm::vec2 tmp_texcoord;

        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside, tmp_uv);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside,tmp_uv);
            }
            else if (geom.type == MESH) 
            {
                t = meshIntersectionTestBVH(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside, tmp_uv, 
                    bvh, meshes, vertices, normals, texcoords, tmp_material_index);
            }
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
                uv = tmp_uv;
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            // The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
            intersections[path_index].uv = uv;
        }
    }
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeMaterial(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        if (intersection.t > 0.0f)
        {
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;
            if (material.emittance > 0.0f) {
                pathSegments[idx].color *= (materialColor * material.emittance);
                pathSegments[idx].remainingBounces = 0;
            }
            else {
                switch (material.procedualTextureID) {
                     case 1: material.color = checkerboard(intersection.uv); break;
                     case 2: material.color = palettes(intersection.uv); break;
                     default: break;
                }
                float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
                glm::vec3 intersect = intersection.t * pathSegments[idx].ray.direction + pathSegments[idx].ray.origin;
                scatterRay(pathSegments[idx], intersect, intersection.surfaceNormal, material, rng);
            }
        }
        else {
            pathSegments[idx].color = glm::vec3(0.0f);
            pathSegments[idx].remainingBounces = 0;
        }
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_intersections
#if BVH
            , dev_bvh
#endif 
            , dev_meshes, dev_vertices, dev_normals, dev_texcoords
        );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

        // --- Shading Stage ---
#ifndef SORT_MATERIAL_ID
        thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, materialsCmp());
#endif
        shadeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials
        );

#ifdef STREAM_COMPACTION
        num_paths = thrust::partition(thrust::device,
            dev_paths, dev_paths + num_paths, isRayAlive()) - dev_paths;
#endif

        iterationComplete = depth == traceDepth || num_paths == 0;

        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

#ifdef STREAM_COMPACTION
    num_paths = dev_path_end - dev_paths;
#endif

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
