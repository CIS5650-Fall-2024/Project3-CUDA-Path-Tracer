#include "hip/hip_runtime.h"
#include "pathtrace.h"

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__ thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

// Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

// implement SampleUniformDiskConcentric from PBRT
__host__ __device__ glm::vec2 sampleUniformDiskConcentric(glm::vec2 u) {
    // Map u to [-1,1]^2
    glm::vec2 uOffset = 2.0f * u - glm::vec2(1.0f);
    if (uOffset.x == 0 && uOffset.y == 0)
        return glm::vec2(0.0f);

    float theta, r;
    if (abs(uOffset.x) > abs(uOffset.y)) {
        r = uOffset.x;
        theta = (PI / 4.0f) * (uOffset.y / uOffset.x);
    } else {
        r = uOffset.y;
        theta = (PI / 2.0f) - (PI / 4.0f) * (uOffset.x / uOffset.y);
    }

    return r * glm::vec2(cos(theta), sin(theta));
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;

static glm::vec3* dev_vertices = NULL;
static glm::vec3* dev_normals = NULL;
static glm::vec2* dev_uvs = NULL;
static Triangle* dev_triangles = NULL;
static LinearBVHNode* dev_linearBVHNodes = NULL;

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_vertices, scene->vertices.size() * sizeof(glm::vec3));
    hipMemcpy(dev_vertices, scene->vertices.data(), scene->vertices.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);

    hipMalloc(&dev_normals, scene->normals.size() * sizeof(glm::vec3));
    hipMemcpy(dev_normals, scene->normals.data(), scene->normals.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);

    hipMalloc(&dev_uvs, scene->uvs.size() * sizeof(glm::vec2));
    hipMemcpy(dev_uvs, scene->uvs.data(), scene->uvs.size() * sizeof(glm::vec2), hipMemcpyHostToDevice);

    hipMalloc(&dev_triangles, scene->triangles.size() * sizeof(Triangle));
    hipMemcpy(dev_triangles, scene->triangles.data(), scene->triangles.size() * sizeof(Triangle), hipMemcpyHostToDevice);

#if USE_BVH
    hipMalloc(&dev_linearBVHNodes, scene->linearBVH.size() * sizeof(LinearBVHNode));
    hipMemcpy(dev_linearBVHNodes, scene->linearBVH.data(), scene->linearBVH.size() * sizeof(LinearBVHNode), hipMemcpyHostToDevice);
#endif

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);

    hipFree(dev_vertices);
    hipFree(dev_normals);
    hipFree(dev_uvs);
    hipFree(dev_triangles);

#if USE_BVH
    hipFree(dev_linearBVHNodes);
#endif

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
        segment.insideObject = false;

        // Stochastic Antialiasing
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
        thrust::uniform_real_distribution<float> u01(0, 1);
        float jitterX = u01(rng) - 0.5f;
        float jitterY = u01(rng) - 0.5f;

        glm::vec3 rayDirection = glm::normalize(cam.view
           - cam.right * cam.pixelLength.x * ((float)x + jitterX - (float)cam.resolution.x * 0.5f)
           - cam.up * cam.pixelLength.y * ((float)y + jitterY - (float)cam.resolution.y * 0.5f)
        );

        // Apply depth of field
        if (cam.aperture > 0.0f) {
            // Sample point on the lens aperture using concentric mapping
            glm::vec2 pLens = cam.aperture * sampleUniformDiskConcentric(glm::vec2(u01(rng), u01(rng)));
            glm::vec3 lensOffset = pLens.x * cam.right + pLens.y * cam.up;

            // Compute focal point
            float t = cam.focalDistance / glm::dot(rayDirection, glm::normalize(cam.view));
            glm::vec3 focalPoint = cam.position + rayDirection * t;

            // Adjust ray origin and direction
            segment.ray.origin = cam.position + lensOffset;
            segment.ray.direction = glm::normalize(focalPoint - segment.ray.origin);
        } else {
            segment.ray.direction = rayDirection;
        }
    }
}

// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    ShadeableIntersection* intersections,
    glm::vec3* vertices,
    glm::vec3* normals,
    glm::vec2* uvs,
    Triangle* triangles,
    LinearBVHNode* linearBVHNodes)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        glm::vec2 uv;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;
        glm::vec2 tmp_uv;
        bool tmp_outside = true;
        int tmp_materialid = -1;

        // naive parse through global geoms
        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, tmp_outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, tmp_outside);

            } else if (geom.type == MESH)
            {
#if USE_BVH
                t = meshIntersectionTestWithLinearBVH(geom,
                                                pathSegment.ray,
                                                tmp_intersect,
                                                tmp_normal,
                                                tmp_uv,
                                                tmp_outside,
                                                vertices,
                                                normals,
                                                uvs,
                                                triangles,
                                                linearBVHNodes,
                                                tmp_materialid);
#else
                t = meshIntersectionTest(geom,
                                         pathSegment.ray,
                                         tmp_intersect,
                                         tmp_normal,
                                         tmp_uv,
                                         tmp_outside,
                                         vertices,
                                         normals,
                                         uvs,
                                         triangles,
                                         tmp_materialid);
#endif
            }

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
                uv = tmp_uv;
                outside = tmp_outside;

                if (geom.type == MESH) {
                    intersections[path_index].materialId = tmp_materialid;
                } else {
                    intersections[path_index].materialId = geoms[hit_geom_index].materialid;
                }
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            // The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].surfaceNormal = normal;
            intersections[path_index].uv = uv;
            intersections[path_index].outside = outside;
            intersections[path_index].intersectionPoint = intersect_point;
        }
    }
}

__global__ void shadeMaterial(
    int iter,
    int depth,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) return;

    // Preload frequently accessed variables into registers
    PathSegment& segment = pathSegments[idx];
    int remainingBounces = segment.remainingBounces;
    if (remainingBounces <= 0) return;

    ShadeableIntersection intersection = shadeableIntersections[idx];

    if (intersection.t > 0.0f) { // If the intersection exists...
        // Set up RNG
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);

        Material material = materials[intersection.materialId];
        glm::vec3 materialColor = material.color;

        if (material.emittance > 0.0f) {
            segment.color *= (materialColor * material.emittance);
            segment.remainingBounces = 0;  // Terminate ray if it hits a light source
        } else {
            glm::vec3 intersectionPoint = intersection.intersectionPoint;
            segment.insideObject = !intersection.outside;

            scatterRay(segment, intersectionPoint, intersection.surfaceNormal, material, rng);

            segment.remainingBounces--;

           if (segment.remainingBounces == 0) {
               segment.color = glm::vec3(0.0f);
           }

            // Apply Russian roulette. Reference to PBRT
#if RUSSIAN_ROULETTE
            // start after a minimum number of bounces
            if (depth > MIN_BOUNCES && segment.remainingBounces > 0) {
                float y = glm::max(glm::max(segment.color.r, segment.color.g), segment.color.b);
                float q = glm::max(MIN_SURVIVAL_PROB, 1.0f - y);

                thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);
                float randVal = u01(rng);

                if (randVal < q) {
                    // Terminate the path
                    segment.color = glm::vec3(0.0f);
                    segment.remainingBounces = 0;
                } else {
                    // Survive
                    segment.color /= (1.0f - q);
                }
            }
#endif
        }
    } else {
        // If no intersection, black out the ray
        segment.color = glm::vec3(0.0f);
        segment.remainingBounces = 0;
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];

        // Clamp the color
        glm::vec3 color = iterationPath.color;
        float maxColorValue = 10.0f;
        color = glm::min(color, glm::vec3(maxColorValue));

        image[iterationPath.pixelIndex] += color;
    }
}

struct IsActive {
    __host__ __device__
    bool operator()(const PathSegment& path) {
        return path.remainingBounces > 0;
    }
};

struct CompareByMaterial {
    __host__ __device__
    bool operator()(const ShadeableIntersection& a, const ShadeableIntersection& b) const {
        return a.materialId < b.materialId;
    }
};

void sortByMaterial(int num_paths) {
    thrust::sort_by_key(
            thrust::device,
            dev_intersections,
            dev_intersections + num_paths,
            dev_paths,
            CompareByMaterial()
    );
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 256;

    ///////////////////////////////////////////////////////////////////////////

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_intersections,
            dev_vertices,
            dev_normals,
            dev_uvs,
            dev_triangles,
            dev_linearBVHNodes
        );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

        // TODO:
        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        //  path segments that have been reshuffled to be contiguous in memory.

        if (SORTMATERIAL) {
            sortByMaterial(num_paths);
        }

        shadeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
            iter,
            depth,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials);

        // Stream compaction
        dev_path_end = thrust::partition(thrust::device, dev_paths, dev_path_end, IsActive());
        num_paths = dev_path_end - dev_paths;

        iterationComplete = depth > traceDepth || num_paths == 0;

        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
