#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"
#include "bvh.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...
static glm::vec3* dev_meshVertices = NULL;
static glm::ivec3* dev_meshFaceIndices = NULL;
static glm::vec3* dev_meshFaceNormals = NULL;
static int* dev_meshFaceMatIndices = NULL;

static bvhNode* dev_bvhNodes = NULL;
static int* dev_meshFaceIndicesBVH = NULL;

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));
    
    // mesh 
    hipMalloc(&dev_meshVertices, scene->mesh.vertices.size() * sizeof(glm::vec3));
    hipMemcpy(dev_meshVertices, scene->mesh.vertices.data(), scene->mesh.vertices.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMalloc(&dev_meshFaceIndices, scene->mesh.faceIndices.size() * sizeof(glm::ivec3));
    hipMemcpy(dev_meshFaceIndices, scene->mesh.faceIndices.data(), scene->mesh.faceIndices.size() * sizeof(glm::ivec3), hipMemcpyHostToDevice);
    hipMalloc(&dev_meshFaceNormals, scene->mesh.faceNormals.size() * sizeof(glm::vec3));
    hipMemcpy(dev_meshFaceNormals, scene->mesh.faceNormals.data(), scene->mesh.faceNormals.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMalloc(&dev_meshFaceMatIndices, scene->mesh.faceMatIndices.size() * sizeof(int));
    hipMemcpy(dev_meshFaceMatIndices, scene->mesh.faceMatIndices.data(), scene->mesh.faceMatIndices.size() * sizeof(int), hipMemcpyHostToDevice);
    // json geom
    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);
    // material
    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);
    // bvh
    hipMalloc(&dev_bvhNodes, sizeof(bvhNode) * scene->bvhNodes.size());
    hipMemcpy(dev_bvhNodes, scene->bvhNodes.data(), scene->bvhNodes.size() * sizeof(bvhNode), hipMemcpyHostToDevice);
    hipMalloc(&dev_meshFaceIndicesBVH, scene->mesh.faceIndicesBVH.size() * sizeof(int));
    hipMemcpy(dev_meshFaceIndicesBVH, scene->mesh.faceIndicesBVH.data(), scene->mesh.faceIndicesBVH.size() * sizeof(int), hipMemcpyHostToDevice);
    
    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created
    hipFree(dev_meshVertices);
    hipFree(dev_meshFaceIndices);
    hipFree(dev_meshFaceNormals);
    hipFree(dev_meshFaceMatIndices);
    hipFree(dev_bvhNodes);
    hipFree(dev_meshFaceIndicesBVH);
    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        // TODO: implement antialiasing by jittering the ray
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
        );

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    glm::vec3* vertices,
    glm::ivec3* faceIndices,
    glm::vec3* faceNormals,
    int* faceMatIndices,
    bvhNode* bvhNodes,
    int* faceIndicesBVH,
    int face_count,
    ShadeableIntersection* intersections,
    bool useBVH)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;
        int material_id = -1;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;

        // naive parse through global geoms

        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            // TODO: add more intersection tests here... triangle? metaball? CSG?

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
                material_id = geom.materialid;
            }
        }
        Ray &ray = pathSegment.ray;
        if (!useBVH){
            for (int i = 0; i < face_count; i++) {
                glm::ivec3& face = faceIndices[i];
                glm::vec3& v0 = vertices[face.x];
                glm::vec3& v1 = vertices[face.y];
                glm::vec3& v2 = vertices[face.z];

                glm::vec3 baryPosition;
                if (glm::intersectRayTriangle(ray.origin, ray.direction, v0, v1, v2, baryPosition)) {
                    t = baryPosition.z;
                    // Check if the ray direction is in the opposite direction as the normal
                    glm::vec3 faceNormal = faceNormals[i];
                    if (t > 0.0f && t < t_min) {
                        t_min = t;
                        hit_geom_index = i;
                        intersect_point = ray.origin + t * ray.direction;
                        normal = faceNormals[i];
                        material_id = faceMatIndices[i];
                    }
                }
            }
        }else{
            bool hit;
            float t = FLT_MAX;
            int faceIndexHit = -1;
            BVHHitTestIterative(
                ray, bvhNodes, 
                vertices, faceIndices, faceNormals, faceIndicesBVH,
                t, faceIndexHit, hit);
            if (t > 0.0f && t < t_min) {
                t_min = t;
                hit_geom_index = faceIndexHit;
                normal = faceNormals[faceIndexHit];
                material_id = faceMatIndices[faceIndexHit];
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
            pathSegments[path_index].remainingBounces = 0;
        }
        else
        {
            // The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = material_id;
            intersections[path_index].surfaceNormal = normal;
        }
    }
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeMaterial(
    int iter,
    int currDepth,
    int maxDepth,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials,
    bool renderWithPathTracing)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        if (intersection.t > 0.0f) // if the intersection exists...
        {
          // Set up the RNG
          // LOOK: this is how you use thrust's RNG! Please look at
          // makeSeededRandomEngine as well.
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;

            if (pathSegments[idx].remainingBounces <= 0) {
                return;
            }
            // If the material indicates that the object was a light, "light" the ray
            if (material.emittance > 0.0f) {
                pathSegments[idx].color *= (materialColor * material.emittance);
                pathSegments[idx].remainingBounces = 0;
            }else if (currDepth < maxDepth) {
            // Otherwise, do some pseudo-lighting computation. This is actually more
            // like what you would expect from shading in a rasterizer like OpenGL.
            // TODO: replace this! you should be able to start with basically a one-liner
                if (renderWithPathTracing){
                    scatterRay(pathSegments[idx], getPointOnRay(pathSegments[idx].ray, intersection.t), intersection.surfaceNormal, material, rng);
                    pathSegments[idx].remainingBounces--;
                }else{
                    float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
                    pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
                    pathSegments[idx].color *= u01(rng); // apply some noise because why not
                    pathSegments[idx].remainingBounces = 0;
                }
            }
            else {
                pathSegments[idx].color = glm::vec3(0.0f);
                pathSegments[idx].remainingBounces = 0;
            }
            // If there was no intersection, color the ray black.
            // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
            // used for opacity, in which case they can indicate "no opacity".
            // This can be useful for post-processing and image compositing.
        }
        else {
            pathSegments[idx].color = glm::vec3(0.0f);
            pathSegments[idx].remainingBounces = 0;
        }
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

struct CompactPathSegments {
    __host__ __device__
    bool operator()(const PathSegment& path) {
        return path.remainingBounces > 0;
    }
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // collision check
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_meshVertices,
            dev_meshFaceIndices,
            dev_meshFaceNormals,
            dev_meshFaceMatIndices,
            dev_bvhNodes,
            dev_meshFaceIndicesBVH,
            hst_scene->mesh.faceIndices.size(),
            dev_intersections,
            hst_scene->useBVH || hst_scene->useBasicBVC
        );
        checkCUDAError("computeIntersections");
        hipDeviceSynchronize();
        depth++;

        // TODO:
        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        // path segments that have been reshuffled to be contiguous in memory.
    
        shadeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
            iter,
            depth,
            traceDepth,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials,
            hst_scene->renderWithPathTracing
        );
        checkCUDAError("shadeMaterial");
        
        dev_path_end = thrust::partition(thrust::device, dev_paths, dev_path_end, CompactPathSegments());
        num_paths = dev_path_end - dev_paths;
        iterationComplete = (depth >= traceDepth || num_paths <= 0);
        checkCUDAError("stream compact");

        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
