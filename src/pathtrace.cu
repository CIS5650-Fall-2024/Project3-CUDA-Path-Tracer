#include "hip/hip_runtime.h"
#pragma once

#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <stack>
#include <thrust/execution_policy.h>
#include <thrust/partition.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <OpenImageDenoise/oidn.hpp>
#if LOG_PERF
#include <fstream>
#endif

#include "bvh.h"
#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"
#include "samplers.h"
#include "flags.h"

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static BVH::Node* dev_nodes = NULL;
static Material* dev_materials = NULL;
static glm::vec4* dev_textures = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;

// OIDN
oidn::DeviceRef device;
static glm::vec3* dev_albedo = NULL;
static glm::vec3* dev_normal = NULL;
static glm::vec3* dev_albedo_norm = NULL;
static glm::vec3* dev_normal_norm = NULL;
static glm::vec3* dev_output = NULL;

std::ofstream streamCompactionLogFile;

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    if (hst_scene->restart) {
        hipMemcpy(dev_image, scene->state.image.data(), scene->state.image.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);
        hst_scene->restart = false;
    }
    else
        hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_nodes, scene->nodes.size() * sizeof(BVH::Node));
    hipMemcpy(dev_nodes, scene->nodes.data(), scene->nodes.size() * sizeof(BVH::Node), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_textures, scene->textures.size() * sizeof(glm::vec4));
    hipMemcpy(dev_textures, scene->textures.data(), scene->textures.size() * sizeof(glm::vec4), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    checkCUDAError("pathtraceInit");

    device = oidn::newDevice(oidn::DeviceType::CUDA);
    device.commit();

    hipMalloc(&dev_albedo, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_albedo, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_normal, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_normal, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_albedo_norm, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_albedo_norm, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_normal_norm, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_normal_norm, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_output, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_output, 0, pixelcount * sizeof(glm::vec3));

#if LOG_PERF
    streamCompactionLogFile.open("streamcompactionlog.txt");
#endif
}

void pathtraceFree()
{
#if LOG_PERF
    streamCompactionLogFile.close();
#endif

    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_nodes);
    hipFree(dev_materials);
    hipFree(dev_textures);
    hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created

    hipFree(dev_albedo);
    hipFree(dev_normal);
    hipFree(dev_albedo_norm);
    hipFree(dev_normal_norm);
    hipFree(dev_output);

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int iterModSamplesX, int iterModSamplesY, float invSampleWidth, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);

        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
        thrust::uniform_real_distribution<float> u01(0, 1);

        PathSegment& segment = pathSegments[index];

        // Antialiasing: jitter rays by [0,1] to generate uniformly random direction distribution per pixel
        // Stratified sampling, shoot jittered ray at (i % (sampleWidth * sampleWidth))-th grid
        glm::vec2 jitter = glm::vec2((iterModSamplesX + u01(rng)) * invSampleWidth, (iterModSamplesY + u01(rng)) * invSampleWidth);
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x - 0.5f + jitter[0] - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y - 0.5f + jitter[1] - (float)cam.resolution.y * 0.5f)
        );

        // Depth of Field, construct a new direction pointing the same direction but from new origin AND at focal length away
        glm::vec2 apertureOrigin = cam.apertureSize * randomOnUnitCircle(rng);
        segment.ray.origin = cam.position + cam.right * apertureOrigin[0] + cam.up * apertureOrigin[1];
        segment.ray.direction = glm::normalize(segment.ray.direction * cam.focalLength + cam.position - segment.ray.origin);

        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);
        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth,
    int numPaths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geomsSize,
    BVH::Node* nodes,
    int nodesSize,
    int rootIdx,
    glm::vec4* textures,
    ShadeableIntersection* intersections)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index >= numPaths) return;

    PathSegment pathSegment = pathSegments[path_index];

    float t;
    glm::vec3 intersect_point;
    glm::vec3 normal;
    glm::vec2 texCoord;
    float t_min = FLT_MAX;
    int hit_geom_index = -1;
    bool outside = true;

    glm::vec3 tmp_intersect;
    glm::vec3 tmp_normal;
    glm::vec2 tmp_texCoord;
    
    // Early terminate if no intersection with the root node
    glm::vec2 times;
    if (bboxIntersectionTest(nodes[rootIdx].bbox, pathSegment.ray, tmp_intersect, tmp_normal, outside, times) < 0.f) {
        intersections[path_index].t = -1.0f;
        intersections[path_index].materialId = -1;
        return;
    }

    // BVH intersection hierarchy
    // Don't render details / far away objects beyond 1024 hierarchical levels
    // 1024 is an arbitrary depth limit since dynamic array sizing is bad
    int nodeStack[1024];
    memset(nodeStack, 0, 1024);
    int nodeStackFinger = 0;

    nodeStack[nodeStackFinger] = rootIdx;
    nodeStackFinger++;

    while (nodeStackFinger > 0 && nodeStackFinger < 1024) {
        int currIdx = nodeStack[nodeStackFinger - 1];
        const BVH::Node& node = nodes[currIdx];
        nodeStackFinger--;

        bool hit = bboxIntersectionTest(node.bbox, pathSegment.ray, tmp_intersect, tmp_normal, outside, times) > 0.f;
        if (!hit || hit && times[0] > t_min) continue;

        if (node.l == node.r) {
            for (int i = node.start; i < node.start + node.size; i++) {
                Geom& geom = geoms[i];
                if (geom.type == CUBE)
                {
                    t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, tmp_texCoord, outside);
                }
                else if (geom.type == SPHERE)
                {
                    t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, tmp_texCoord, outside);
                }
                else if (geom.type == TRIANGLE)
                {
                    t = triangleIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, tmp_texCoord, textures, outside);
                }

                // Compute the minimum t from the intersection tests to determine what
                // scene geometry object was hit first.
                if (t > 0.0f && t_min > t)
                {
                    t_min = t;
                    hit_geom_index = i;
                    intersect_point = tmp_intersect;
                    normal = tmp_normal;
                    texCoord = tmp_texCoord;
                }
            }
            continue;
        }
      
        // Check intersection with left and right children
        bool hitL = bboxIntersectionTest(nodes[node.l].bbox, pathSegment.ray, tmp_intersect, tmp_normal, outside, times) > 0.f;
        bool hitR = bboxIntersectionTest(nodes[node.r].bbox, pathSegment.ray, tmp_intersect, tmp_normal, outside, times) > 0.f;

        if (hitL && hitR) {
            // Both hit
            nodeStack[nodeStackFinger] = node.l;
            nodeStackFinger++;
            nodeStack[nodeStackFinger] = node.r;
            nodeStackFinger++;
        } else if (hitR) {
            nodeStack[nodeStackFinger] = node.r;
            nodeStackFinger++;
        } else if (hitL) {
            nodeStack[nodeStackFinger] = node.l;
            nodeStackFinger++;
        }
    }

    if (hit_geom_index == -1)
    {
        intersections[path_index].t = -1.0f;
        intersections[path_index].materialId = -1;
    }
    else
    {
        // The ray hits something
        intersections[path_index].t = t_min;
        intersections[path_index].materialId = geoms[hit_geom_index].materialid;
        intersections[path_index].surfaceNormal = normal;
        intersections[path_index].texCoord = texCoord;
    }
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeMaterial(
    int iter,
    int numPaths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials,
    glm::vec4* textures,
    ImageTextureInfo bgTextureInfo,
    glm::vec3* dev_img,
    glm::vec3* albedos,
    glm::vec3* normals,
    int depth)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numPaths) return;
    if (pathSegments[idx].remainingBounces < 0) return;

    // Set up the RNG
    thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);

    // Scatter the ray at intersecting point and perform bsdf evaluation
    scatterRay(pathSegments[idx], 
        pathSegments[idx].ray.origin + 
        pathSegments[idx].ray.direction * shadeableIntersections[idx].t, 
        shadeableIntersections[idx],
        materials[shadeableIntersections[idx].materialId],
        textures,
        bgTextureInfo,
        rng,
        dev_img,
        albedos,
        normals,
        depth);
}

__global__ void averageOIDNArrays(
    int iter,
    int numPaths,
    glm::vec3* albedos,
    glm::vec3* normals,
    glm::vec3* albedos_norm,
    glm::vec3* normals_norm)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numPaths) return;

    albedos_norm[idx] = albedos[idx] / (float)iter;
    normals_norm[idx] = glm::normalize(normals[idx]);
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;
    const int sampleWidth = hst_scene->state.sampleWidth;
    const float invSampleWidth = 1.0f / (float)sampleWidth;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

    int iterModSamples = iter % (sampleWidth * sampleWidth);
    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, 
                                                            iterModSamples % sampleWidth, 
                                                            iterModSamples / sampleWidth, 
                                                            invSampleWidth, 
                                                            traceDepth, 
                                                            dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    ShadeableIntersection* dev_intersections_end = dev_intersections + pixelcount;
    int remaining_paths = pixelcount;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    // Terminate iteration once maximum trace depth has been reached or if no valid rays remain
    while (remaining_paths && depth < traceDepth)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, remaining_paths * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (remaining_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
            depth,
            remaining_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_nodes,
            hst_scene->nodes.size(),
            hst_scene->bvhRootIdx,
            dev_textures,
            dev_intersections
        );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

#if PATHTRACE_CONTIGUOUS_MATERIALID
        dev_intersections_end = dev_intersections + remaining_paths;
        // Sort arrays in decreasing materialId order and contiguous memory
        // We do not terminate paths here as we need to apply environmet mapping
        thrust::stable_sort_by_key(
            thrust::device, 
            dev_intersections, 
            dev_intersections_end, 
            dev_paths,
            [] __device__(const ShadeableIntersection & si1, const ShadeableIntersection & si2) { return si1.materialId > si2.materialId; });
#endif

        // TODO:
        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        // path segments that have been reshuffled to be contiguous in memory.

        shadeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
            iter,
            remaining_paths,
            dev_intersections,
            dev_paths,
            dev_materials,
            dev_textures,
            hst_scene->bgTextureInfo,
            dev_image,
            dev_albedo,
            dev_normal,
            depth
        );
        checkCUDAError("shade material error");

#if STREAM_COMPACTION
        // Compaction : Terminate paths with no more remaining bounces
        dev_path_end = thrust::stable_partition(
            thrust::device, 
            dev_paths, 
            dev_path_end, 
            [] __device__ (const PathSegment& ps) { return ps.remainingBounces > -1; });

        remaining_paths = dev_path_end - dev_paths;
#endif

#if LOG_PERF
        streamCompactionLogFile << remaining_paths << "\n";
#endif

        if (guiData)
        {
            guiData->TracedDepth = depth;
        }
    }

    ///////////////////////////////////////////////////////////////////////////

#if USE_OIDN
    // Perform basic denoising for the real time renders for performance
    // Based on https://github.com/RenderKit/oidn?tab=readme-ov-file#basic-denoising-c11-api
    
    // Normalize albedo and normal arrays (currently summed up iter times)
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    averageOIDNArrays << <numBlocksPixels, blockSize1d >> > (iter, pixelcount, dev_albedo, dev_normal, dev_albedo_norm, dev_normal_norm);

    // Create a filter for denoising a beauty (color) image using optional auxiliary images too
    // This can be an expensive operation, so try no to create a new filter for every image!
    oidn::FilterRef filter = device.newFilter("RT"); // generic ray tracing filter
    filter.setImage("color",  dev_image,  oidn::Format::Float3, cam.resolution.x, cam.resolution.y); // beauty
    filter.setImage("albedo", dev_albedo_norm, oidn::Format::Float3, cam.resolution.x, cam.resolution.y); // auxiliary
    filter.setImage("normal", dev_normal_norm, oidn::Format::Float3, cam.resolution.x, cam.resolution.y); // auxiliary
    filter.setImage("output", dev_output, oidn::Format::Float3, cam.resolution.x, cam.resolution.y); // denoised beauty
    filter.set("hdr", true); // beauty image is HDR
    filter.commit();

    // Filter the beauty image
    filter.execute();

    // Check for errors
    const char* errorMessage;
    if (device.getError(errorMessage) != oidn::Error::None)
        std::cout << "Error: " << errorMessage << std::endl;

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_output);
#else
    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);
#endif
}

void retrieveRenderBuffer() {
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

#if USE_OIDN
    // Perform denoising with prefiltering for the saved images
    // Based on https://github.com/RenderKit/oidn?tab=readme-ov-file#denoising-with-prefiltering-c11-api
    
    // Create a filter for denoising a beauty (color) image using prefiltered auxiliary images too
    oidn::FilterRef filter = device.newFilter("RT"); // generic ray tracing filter
    filter.setImage("color",  dev_image,  oidn::Format::Float3, cam.resolution.x, cam.resolution.y); // beauty
    filter.setImage("albedo", dev_albedo_norm, oidn::Format::Float3, cam.resolution.x, cam.resolution.y); // auxiliary
    filter.setImage("normal", dev_normal_norm, oidn::Format::Float3, cam.resolution.x, cam.resolution.y); // auxiliary
    filter.setImage("output", dev_output, oidn::Format::Float3, cam.resolution.x, cam.resolution.y); // denoised beauty
    filter.set("hdr", true); // beauty image is HDR
    filter.set("cleanAux", true); // auxiliary images will be prefiltered
    filter.commit();

    // Create a separate filter for denoising an auxiliary albedo image (in-place)
    oidn::FilterRef albedoFilter = device.newFilter("RT"); // same filter type as for beauty
    albedoFilter.setImage("albedo", dev_albedo_norm, oidn::Format::Float3, cam.resolution.x, cam.resolution.y);
    albedoFilter.setImage("output", dev_albedo_norm, oidn::Format::Float3, cam.resolution.x, cam.resolution.y);
    albedoFilter.commit();

    // Create a separate filter for denoising an auxiliary normal image (in-place)
    oidn::FilterRef normalFilter = device.newFilter("RT"); // same filter type as for beauty
    normalFilter.setImage("normal", dev_normal_norm, oidn::Format::Float3, cam.resolution.x, cam.resolution.y);
    normalFilter.setImage("output", dev_normal_norm, oidn::Format::Float3, cam.resolution.x, cam.resolution.y);
    normalFilter.commit();

    // Prefilter the auxiliary images
    albedoFilter.execute();
    normalFilter.execute();

    // Filter the beauty image
    filter.execute();

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_output,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
#else
    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
#endif

    checkCUDAError("pathtrace");
}

void resetRenderBuffer() {
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));
}