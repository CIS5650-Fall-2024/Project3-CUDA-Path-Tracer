#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"
#include "mathUtils.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];
        pix /= iter;
        pix = math::ACESMapping(pix);
        pix = math::gammaCorrect(pix);

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

__device__ void spawnRay(Ray& ray, const glm::vec3& ori, const glm::vec3& dir)
{
    ray.origin = ori + EPSILON * dir;
    ray.direction = dir;
}

__global__ void getFocalDistance(SceneDev* scene, Camera* cam, float xPos, float yPos)
{
    Ray r;
    ShadeableIntersection isect;
    xPos -= (float)cam->resolution.x * 0.5f;
    yPos -= (float)cam->resolution.y * 0.5f;
    cam->generateRay(r, xPos, yPos);
    scene->intersect(r, isect);
    if (isect.t > 0.f) cam->focalLength = isect.t;
}

__global__ void generateGbuffer(SceneDev* scene, Material* materials, Camera cam, glm::vec3* albedo, glm::vec3* normal)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {

        // gen rays
        int index = x + (y * cam.resolution.x);
        PathSegment segment;

        thrust::default_random_engine rng = makeSeededRandomEngine(0, index, 0);
        thrust::uniform_real_distribution<float> u01(0, 1);

        segment.throughput = glm::vec3(1.0f, 1.0f, 1.0f);
        segment.radiance = glm::vec3(0.f);

        glm::vec2 offset = glm::vec2(u01(rng) - 0.5f, u01(rng) - 0.5f);
        float xPix = (float)x - (float)cam.resolution.x * 0.5f + offset.x;
        float yPix = (float)y - (float)cam.resolution.y * 0.5f + offset.y;
        //cam.generateRayLens(segment.ray, xPix, yPix, u01(rng), u01(rng));
        cam.generateRay(segment.ray, xPix, yPix);

        segment.pixelIndex = index;
        
        // do intersection
        ShadeableIntersection isect;
        scene->intersect(segment.ray, isect);

        // if no hit event, sample env map
        if (isect.t < 0.f)
        {
            albedo[index] = scene->getEnvColor(segment.ray.direction);
            normal[index] = glm::vec3(0);
        }
        else
        {
            Material material = materials[isect.materialId];
            material.createMaterialInst(material, isect.uv);
            albedo[index] = material.albedo;
            normal[index] = isect.nor;
        }

    }
}

struct CompactPaths
{
    __host__ __device__ bool operator() (const PathSegment& segment)
    {
        return segment.remainingBounces == 0;
    }
};

struct CompactIsects
{
    __host__ __device__ bool operator() (const ShadeableIntersection& isect)
    {
        return isect.t < 0.f;
    }
};

struct CopyFinishedPaths
{
    __host__ __device__ bool operator() (const PathSegment& segment)
    {
        return segment.remainingBounces != 0;
    }
};

struct SortPathByKey
{
    __host__ __device__ bool operator() (const ShadeableIntersection& isect1, const ShadeableIntersection& isect2)
    {
        return isect1.materialId < isect2.materialId;
    }
};

static Scene* hst_scene = NULL;
static SceneDev* sceneDev = NULL;
static SceneDev* dev_sceneDev = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static glm::vec3* dev_albedoBuffer = NULL;
static glm::vec3* dev_normalBuffer = NULL;
static Camera* dev_cam = NULL;

static PathSegment* dev_paths = NULL;
static thrust::device_ptr<PathSegment> dev_paths_thrust;

static PathSegment* dev_paths_finish = NULL;
static thrust::device_ptr<PathSegment> dev_paths_finish_thrust;

static ShadeableIntersection* dev_intersections = NULL;
static thrust::device_ptr<ShadeableIntersection> dev_intersections_thrust;


void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_albedoBuffer, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_albedoBuffer, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_normalBuffer, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_normalBuffer, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));
    dev_paths_thrust = thrust::device_ptr<PathSegment>(dev_paths);

    hipMalloc(&dev_paths_finish, pixelcount * sizeof(PathSegment));
    dev_paths_finish_thrust = thrust::device_ptr<PathSegment>(dev_paths_finish);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
    dev_intersections_thrust = thrust::device_ptr<ShadeableIntersection>(dev_intersections);

    sceneDev = scene->sceneDev;
    hipMalloc(&dev_sceneDev, sizeof(SceneDev));
    hipMemcpy(dev_sceneDev, sceneDev, sizeof(SceneDev), hipMemcpyHostToDevice);

    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    generateGbuffer << <blocksPerGrid2d, blockSize2d >> > (dev_sceneDev, sceneDev->materials, cam, dev_albedoBuffer, dev_normalBuffer);

    hipMemcpy(hst_scene->state.albedo.data(), dev_albedoBuffer,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    hipMemcpy(hst_scene->state.normal.data(), dev_normalBuffer,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);


    // auto focus
    hipMalloc(&dev_cam, sizeof(Camera));
    hipMemcpy(dev_cam, &scene->state.camera, sizeof(Camera), hipMemcpyHostToDevice);
    getFocalDistance << <1, 1>> > (dev_sceneDev, dev_cam, scene->mouseClickPos.x, scene->mouseClickPos.y);
    hipMemcpy(&scene->state.camera, dev_cam, sizeof(Camera), hipMemcpyDeviceToHost);
    hipFree(dev_cam);
    std::printf("new focal distance: %f at: %f, %f\n", scene->state.camera.focalLength, scene->mouseClickPos.x, scene->mouseClickPos.y);

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);
    hipFree(dev_albedoBuffer);
    hipFree(dev_normalBuffer);
    hipFree(dev_paths);
    hipFree(dev_paths_finish);
    hipFree(dev_intersections);
    hipFree(dev_sceneDev);
    // TODO: clean up any extra device memory you created

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, traceDepth);
        thrust::uniform_real_distribution<float> u01(0, 1);

        segment.throughput = glm::vec3(1.0f, 1.0f, 1.0f);
        segment.radiance = glm::vec3(0.f);

        glm::vec2 offset = glm::vec2(u01(rng) - 0.5f, u01(rng) - 0.5f);
        float xPix = (float)x - (float)cam.resolution.x * 0.5f + offset.x;
        float yPix = (float)y - (float)cam.resolution.y * 0.5f + offset.y;
        cam.generateRayLens(segment.ray, xPix, yPix, u01(rng), u01(rng));
        //cam.generateRay(segment.ray, (float)x - (float)cam.resolution.x * 0.5f + offset.x,
        //    (float)y - (float)cam.resolution.y * 0.5f + offset.y);

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}


__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    SceneDev* scene,
    ShadeableIntersection* intersections)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment& segment = pathSegments[path_index];
        ShadeableIntersection isect;

        scene->intersect(segment.ray, isect);
        intersections[path_index] = isect;

        if (isect.t < 0.f)
        {
            glm::vec3 skyColor = scene->getEnvColor(segment.ray.direction);
            segment.radiance += segment.throughput * skyColor;
            segment.remainingBounces = 0;
            return;
        }

        //float t;
        //glm::vec3 intersect_point;
        //glm::vec3 normal;
        //float t_min = FLT_MAX;
        //uint32_t hit_geom_index = UINT32_MAX;
        //int materialID = -1;
        //bool outside = true;

        //glm::vec3 tmp_intersect;
        //glm::vec3 tmp_normal;

        //// naive parse through global geoms
        //for (uint32_t i = 0; i < scene->primNum; ++i)
        //{
        //    uint32_t primID = scene->primitives[i].primId;
        //    if (primID < scene->triNum)
        //    {
        //        glm::vec3 bary;
        //        t = triangleIntersection(pathSegment.ray,
        //            scene->vertices[3 * primID], scene->vertices[3 * primID + 1], scene->vertices[3 * primID + 2], tmp_normal, bary);
        //        if (t < 0.f) continue;

        //        if (t > 0.f && t_min > t)
        //        {
        //            t_min = t;
        //            hit_geom_index = primID;
        //            materialID = scene->primitives[i].materialId;
        //            intersect_point = pathSegment.ray.origin + t * pathSegment.ray.direction;
        //            normal = scene->normals[3 * primID] * bary.x + scene->normals[3 * primID + 1] * bary.y
        //                + scene->normals[3 * primID + 2] * bary.z;
        //        }
        //    }
        //    else
        //    {
        //        Geom& geom = scene->geoms[primID - scene->triNum];

        //        if (geom.type == CUBE)
        //        {
        //            t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
        //        }
        //        else if (geom.type == SPHERE)
        //        {
        //            t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
        //        }

        //        if (t > 0.0f && t_min > t)
        //        {
        //            t_min = t;
        //            hit_geom_index = primID;
        //            materialID = scene->primitives[i].materialId;
        //            intersect_point = tmp_intersect;
        //            normal = tmp_normal;
        //        }

        //    }
        //}

        //intersections[path_index].primId = hit_geom_index;
        //if (hit_geom_index == UINT32_MAX)
        //{
        //    intersections[path_index].t = -1.0f;
        //}
        //else
        //{
        //    // The ray hits something
        //    intersections[path_index].t = t_min;
        //    intersections[path_index].materialId = materialID;
        //    intersections[path_index].nor = normal;
        //}
    }
}

__global__ void sampleSurface(
    int iter,
    int num_paths,
    SceneDev* scene,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) {
        return;
    }

    ShadeableIntersection isect = shadeableIntersections[idx];
    PathSegment& segment = pathSegments[idx];
    glm::vec3 hitPoint = segment.ray.origin + isect.t * segment.ray.direction;

    // case when ray hit nothing
    /*
    if (isect.primId == UINT32_MAX)
    {
        scene->sampleEnv(segment);
        segment.remainingBounces = 0;
        return;
    }
    else
    {
        segment.radiance = glm::vec3(isect.t / 100.f);
        segment.remainingBounces = 0;
        return;
    }
    */

    thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, segment.remainingBounces);
    thrust::uniform_real_distribution<float> u01(0, 1);

    Material material = materials[isect.materialId];
    material.createMaterialInst(material, isect.uv);

    // case we have a light hit
    if (material.type == Light)
    {
        segment.remainingBounces = 0;
        segment.throughput *= material.albedo * material.emittance;
        segment.radiance += segment.throughput;
    }
    else
    {
        glm::vec3 rn = glm::vec3(u01(rng), u01(rng), u01(rng));
        float absCos;
        float pdf = 1.f;

        // do a light sample
        float liPdf;
        float lightWeight = 0.f;
        glm::vec3 wi;
        glm::vec3 radiance = scene->sampleEnv(hitPoint, wi, rn, &liPdf);
        if (liPdf > EPSILON)
        {
            absCos = math::clampDot(wi, isect.nor);
            glm::vec3 f = material.getBSDF(isect.nor, segment.ray.direction, wi, &pdf);
            radiance = f * radiance * absCos / liPdf;
            lightWeight = math::powerHeuristic(liPdf, pdf);
            segment.radiance += lightWeight * (segment.throughput * radiance);
        }

        // bsdf sample
        wi = glm::vec3(0.f);
        pdf = 1.f;
        glm::vec3 bsdf = material.samplef(isect.nor, segment.ray.direction, wi, rn, &pdf);

        if (pdf < EPSILON)
        {
            segment.remainingBounces = 0;
        }
        else
        {
            absCos = (material.type == Specular) ? 1.f : math::clampDot(wi, isect.nor);
            segment.throughput *= bsdf * (absCos / pdf);
            spawnRay(segment.ray, hitPoint, wi);
            --segment.remainingBounces;
        }
    }


    
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.radiance;
    }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    thrust::device_ptr<PathSegment> finished_tail = dev_paths_finish_thrust;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_sceneDev,
            dev_intersections
        );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

        finished_tail = thrust::remove_copy_if(dev_paths_thrust, dev_paths_thrust + num_paths, finished_tail, CopyFinishedPaths());
        thrust::remove_if(dev_intersections_thrust, dev_intersections_thrust + num_paths, CompactIsects());
        auto arrTail = thrust::remove_if(dev_paths_thrust, dev_paths_thrust + num_paths, CompactPaths());
        num_paths = arrTail - dev_paths_thrust;
        
        if (num_paths == 0) break;
        // sort rays
        //thrust::sort_by_key(dev_intersections_thrust, dev_intersections_thrust + num_paths, dev_paths_thrust, SortPathByKey());

        numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        sampleSurface<<<numblocksPathSegmentTracing, blockSize1d>>>(
            iter,
            num_paths,
            dev_sceneDev,
            dev_intersections,
            dev_paths,
            sceneDev->materials
        );

        finished_tail = thrust::remove_copy_if(dev_paths_thrust, dev_paths_thrust + num_paths, finished_tail, CopyFinishedPaths());
        arrTail = thrust::remove_if(dev_paths_thrust, dev_paths_thrust + num_paths, CompactPaths());
        num_paths = arrTail - dev_paths_thrust;

        iterationComplete = (num_paths == 0);

        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

    // Assemble this iteration and apply it to the image
    num_paths = finished_tail - dev_paths_finish_thrust;
    dim3 numBlocksPixels = (num_paths + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths_finish);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
