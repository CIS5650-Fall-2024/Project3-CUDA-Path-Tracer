#include "hip/hip_runtime.h"
#include "bvh.h"

// BBox class
BBox::BBox() : minC(glm::vec3(INF_F)), maxC(glm::vec3(-INF_F))
{
	extent = maxC - minC;
}

BBox::BBox(const glm::vec3& p) : minC(p), maxC(p)
{
	extent = maxC - minC;
}

BBox::BBox(const glm::vec3 min, const glm::vec3 max) : minC(min), maxC(max), extent(max - min) {}

BBox::BBox(const BBox& bbox) : minC(bbox.minC), maxC(bbox.maxC), extent(extent) {}

BBox& BBox::operator=(const BBox& bbox)
{
	minC = bbox.minC;
	maxC = bbox.maxC;
	extent = bbox.extent;

	return *this;
}

void BBox::expand(const BBox& bbox) 
{
	minC.x = std::min(minC.x, bbox.minC.x);
	minC.y = std::min(minC.y, bbox.minC.y);
	minC.z = std::min(minC.z, bbox.minC.z);
	maxC.x = std::max(maxC.x, bbox.maxC.x);
	maxC.y = std::max(maxC.y, bbox.maxC.y);
	maxC.z = std::max(maxC.z, bbox.maxC.z);
	extent = maxC - minC;
}

void BBox::expand(const glm::vec3& p)
{
	minC.x = std::min(minC.x, p.x);
	minC.y = std::min(minC.y, p.y);
	minC.z = std::min(minC.z, p.z);
	maxC.x = std::max(maxC.x, p.x);
	maxC.y = std::max(maxC.y, p.y);
	maxC.z = std::max(maxC.z, p.z);
	extent = maxC - minC;
}

glm::vec3 BBox::centroid() const
{
	return (minC + maxC) / 2.0f;
}

float BBox::surfaceArea() const
{
	if (empty())
	{
		return 0.0f;
	}
	else 
	{
		return 2 * (extent.x * extent.z + extent.x * extent.y + extent.y * extent.z);
	}
}

bool BBox::empty() const
{
	return minC.x > maxC.x || minC.y > maxC.y || minC.z > maxC.z;
}

__device__ bool BBox::intersect(const Ray& r, double& t0, double& t1) const
{
	glm::vec3 testMinC = minC;
	double tXMin = (minC.x - r.origin.x) / r.direction.x;
	double tXMax = (maxC.x - r.origin.x) / r.direction.x;
	if (tXMax < tXMin)
	{
		double tmp = tXMin;
		tXMin = tXMax;
		tXMax = tmp;
	}

	double tYMin = (minC.y - r.origin.y) / r.direction.y;
	double tYMax = (maxC.y - r.origin.y) / r.direction.y;
	if (tYMax < tYMin)
	{
		double tmp = tYMin;
		tYMin = tYMax;
		tYMax = tmp;
	}

	double tZMin = (minC.z - r.origin.z) / r.direction.z;
	double tZMax = (maxC.z - r.origin.z) / r.direction.z;
	if (tZMax < tZMin)
	{
		double tmp = tZMin;
		tZMin = tZMax;
		tZMax = tmp;
	}

	double tmin = fmaxf(fmaxf(tXMin, tYMin), tZMin);
	double tmax = fminf(fminf(tXMax, tYMax), tZMax);

	if (tmin > tmax)
	{
		return false;
	}

	if ((tmin >= t0) && (tmax <= t1))
	{
		t0 = tmin;
		t1 = tmax;
	}

	return (t0 < r.tmax) && (t1 > r.tmin);
}

// BVHNode class
BVHNode::BVHNode(BBox bbox) : bb(bbox), leftNodeIndex(-1), rightNodeIndex(-1), primsIndices(nullptr), numPrims(-1) {}

BVHNode::BVHNode(const BVHNode& node) 
{
	bb = node.bb;
	leftNodeIndex = node.leftNodeIndex;
	rightNodeIndex = node.rightNodeIndex;
	numPrims = node.numPrims;
	primsIndices = nullptr;
	
	if (numPrims > 0)
	{
		primsIndices = new int[numPrims];
		std::copy(node.primsIndices, node.primsIndices + numPrims, primsIndices);
	}
}

BVHNode& BVHNode::operator=(const BVHNode& node)
{
	if (this != &node) 
	{
		if (numPrims > 0) 
		{
			delete[] primsIndices;
		}

		bb = node.bb;
		leftNodeIndex = node.leftNodeIndex;
		rightNodeIndex = node.rightNodeIndex;
		numPrims = node.numPrims;

		if (numPrims > 0)
		{
			primsIndices = new int[numPrims];
			std::copy(node.primsIndices, node.primsIndices + numPrims, primsIndices);
		}
	}

	return *this;
}

BVHNode::~BVHNode()
{
	if (numPrims > 0) 
	{
		delete[] primsIndices;
	}
}

void BVHNode::setPrims(const std::vector<int>& pI) 
{
	numPrims = pI.size();
	primsIndices = new int[numPrims];
	std::copy(pI.begin(), pI.end(), primsIndices);
}

__device__ bool BVHNode::isLeaf() const
{
	return numPrims > 0;
}

// BVH Construction Helper
BBox getBBox(const Primitive& prim) 
{
	if (prim.type == TRIANGLE) 
	{
		BBox bbox(prim.p1);
		bbox.expand(prim.p2);
		bbox.expand(prim.p3);
		return bbox;
	}
	else if (prim.type == SPHEREP)
	{
		return BBox(prim.p2, prim.p3);
	}
	else if (prim.type == CUBEP)
	{
		return BBox(prim.p2, prim.p3);
	}
}

int findSplitAxis(glm::vec3 cen) 
{
	int axis;

	if (cen.x >= cen.y)
	{
		if (cen.x >= cen.z)
		{
			axis = 0;
		}
		else
		{
			axis = 2;
		}
	}
	else
	{
		if (cen.y >= cen.z)
		{
			axis = 1;
		}
		else
		{
			axis = 2;
		}
	}

	return axis;
}

// Construct the BVH for the primitives with indices in primsIndices, store the BVHNodes in bvhs, and return the index of the root BVHNode
int constructBVH(const std::vector<Primitive>& prims, const std::vector<int>& primsIndices, std::vector<BVHNode>& bvh, size_t maxLeafSize)
{
	// Populate all primitives
	BBox cetroidBox, bbox;
	for (int i : primsIndices)
	{
		BBox bb = getBBox(prims[i]);
		bbox.expand(bb);
		cetroidBox.expand(bb.centroid());
	}

	// Construct root BVHNode
	BVHNode node(bbox);
	int curIndex = bvh.size();
	bvh.push_back(node);

	if (primsIndices.size() <= maxLeafSize)
	{
		bvh[curIndex].setPrims(primsIndices);
	}
	else 
	{
		int axis = findSplitAxis(cetroidBox.extent);
		float splitPoint = cetroidBox.centroid()[axis];

		double left = 0, right = 0;
		std::vector<int> leftIndices, rightIndices;

		while (leftIndices.empty() || rightIndices.empty()) 
		{
			for (int j : primsIndices)
			{
				float pCentroidAxis = getBBox(prims[j]).centroid()[axis];

				if (pCentroidAxis >= splitPoint) 
				{
					rightIndices.push_back(j);
					right += pCentroidAxis;
				}
				else 
				{
					leftIndices.push_back(j);
					left += pCentroidAxis;
				}
			}

			if (leftIndices.empty()) 
			{
				splitPoint = right / rightIndices.size();
				right = 0;
				rightIndices.clear();
			}
			else if (rightIndices.empty())
			{
				splitPoint = left / leftIndices.size();
				left = 0;
				leftIndices.clear();
			}
			else 
			{
				bvh[curIndex].rightNodeIndex = constructBVH(prims, rightIndices, bvh);
				bvh[curIndex].leftNodeIndex = constructBVH(prims, leftIndices, bvh);
			}
		}
	}
	return curIndex;
}

__device__ bool intersectBVH(Ray& ray, ShadeableIntersection& intersection, Geom* geoms, Primitive* prims, BVHNode* bvh, int cur)
{
	BVHNode& node = bvh[cur];
	int testl = node.leftNodeIndex;
	double t0 = ray.tmin, t1 = ray.tmax;
	if (!node.bb.intersect(ray, t0, t1))
	{
		return false;
	}

	if (node.isLeaf()) 
	{
		bool hit = false;
		for (int i = 0; i < node.numPrims; i++)
		{
			Primitive& p = prims[node.primsIndices[i]];
			bool curHit = false;

			if (p.type == TRIANGLE)
			{
				curHit = triangleIntersectionTest(geoms[p.geomId], p, ray, intersection);
			}
			else if (p.type == CUBEP)
			{
				curHit = boxIntersectionTest(geoms[p.geomId], p, ray, intersection);
			}
			else if (p.type == SPHEREP)
			{
				curHit = sphereIntersectionTest(geoms[p.geomId], p, ray, intersection);
			}

			if (curHit)
			{
				hit = true;
				intersection.primitiveId = node.primsIndices[i];
				intersection.materialId = geoms[p.geomId].materialid;
			}
		}
		return hit;
	}
	else 
	{
		bool interL = intersectBVH(ray, intersection, geoms, prims, bvh, node.leftNodeIndex);
		bool interR = intersectBVH(ray, intersection, geoms, prims, bvh, node.rightNodeIndex);
		return interL || interR;
	}
}