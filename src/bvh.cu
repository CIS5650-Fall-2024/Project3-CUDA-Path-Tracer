#include "hip/hip_runtime.h"
﻿#include "bvh.h"
LinearBVHNode* dev_nodes = NULL;

void BVHAccel::updateMortonCodes(std::vector<MortonPrimitive>& mortonPrims, const std::vector<BVHPrimitiveInfo>& primitiveInfo, AABB& bounds, int chunkSize) const
{
	for (int i = 0; i < mortonPrims.size(); i++)
	{
		// << Update mortonPrims[i] for ith primitive >>
		constexpr int mortonBits = 10; // use 10 bits for each spatial dimension: x, y, z
		constexpr int mortonScale = 1 << mortonBits;
		mortonPrims[i].primitiveIndex = primitiveInfo[i].primitiveNumber;
		glm::vec3 centroidOffset = bounds.Offset(primitiveInfo[i].centroid);
		mortonPrims[i].mortonCode = EncodeMorton3(centroidOffset * static_cast<float>(mortonScale));
	}
}

BVHAccel::BVHBuildNode* BVHAccel::emitLBVH(BVHBuildNode*& buildNodes,
	const std::vector<BVHPrimitiveInfo>& primitiveInfo,
	MortonPrimitive* mortonPrims, int nPrimitives, int* totalNodes,
	std::vector<Triangle*>& orderedPrims,
	std::atomic<int>* orderedPrimsOffset, int bitIndex, int recursionDepth) const
{
	//printf("Recursion depth: %d\n", recursionDepth);
	recursionDepth++;
	if (bitIndex == -1 || nPrimitives < maxPrimsInNode) {
		// Create and return leaf node of LBVH treelet
		(*totalNodes)++;
		BVHBuildNode* node = buildNodes++;
		AABB bounds;
		int firstPrimOffset = orderedPrimsOffset->fetch_add(nPrimitives);
		for (int i = 0; i < nPrimitives; ++i) {
			int primitiveIndex = mortonPrims[i].primitiveIndex;
			orderedPrims[firstPrimOffset + i] = primitives[primitiveIndex];
			bounds = AABB::Union(bounds, primitiveInfo[primitiveIndex].bounds);
		}
		node->initLeaf(firstPrimOffset, nPrimitives, bounds);
		return node;
	}
	else {
		int mask = 1 << bitIndex;
		// Advance to next subtree level if there’s no LBVH split for this bit 
		if ((mortonPrims[0].mortonCode & mask) ==
			(mortonPrims[nPrimitives - 1].mortonCode & mask))
			return emitLBVH(buildNodes, primitiveInfo, mortonPrims, nPrimitives,
				totalNodes, orderedPrims, orderedPrimsOffset,
				bitIndex - 1, recursionDepth);

		// Find LBVH split point for this dimension 
		int searchStart = 0, searchEnd = nPrimitives - 1;
		while (searchStart + 1 != searchEnd) {
			int mid = (searchStart + searchEnd) / 2;
			if ((mortonPrims[searchStart].mortonCode & mask) ==
				(mortonPrims[mid].mortonCode & mask))
				searchStart = mid;
			else
				searchEnd = mid;
		}
		int splitOffset = searchEnd;

		// Create and return interior LBVH node
		(*totalNodes)++;
		BVHBuildNode* node = buildNodes++;
		BVHBuildNode* lbvh[2] = {
			emitLBVH(buildNodes, primitiveInfo, mortonPrims, splitOffset,
					 totalNodes, orderedPrims, orderedPrimsOffset, bitIndex - 1, recursionDepth),
			emitLBVH(buildNodes, primitiveInfo, &mortonPrims[splitOffset],
					 nPrimitives - splitOffset, totalNodes, orderedPrims,
					 orderedPrimsOffset, bitIndex - 1, recursionDepth)
		};
		int axis = bitIndex % 3;
		node->InitInterior(axis, lbvh[0], lbvh[1]);
		return node;

	}
}



int BVHAccel::flattenBVHTree(BVHBuildNode* node, int* offset, int maxNodeNumber) {
	LinearBVHNode* linearNode = &nodes[*offset];

	linearNode->bounds = node->bounds;
	int myOffset = (*offset)++;

	if (node->nPrimitives > 0) {
		linearNode->primitivesOffset = node->firstPrimOffset;
		linearNode->nPrimitives = node->nPrimitives;
	}
	else {
		//Create interior flattened BVH node
		linearNode->axis = node->splitAxis;
		linearNode->nPrimitives = 0;
		if (node && node->children[0] && node->children[1]) {
			flattenBVHTree(node->children[0], offset, maxNodeNumber);
			linearNode->secondChildOffset = flattenBVHTree(node->children[1], offset, maxNodeNumber);
		}
	}
	return myOffset;
}

void BVHAccel::RadixSort(std::vector<MortonPrimitive>* v) {
	std::vector<MortonPrimitive> tempVector(v->size());
	constexpr int bitsPerPass = 6;
	constexpr int nBits = 30;
	constexpr int nPasses = nBits / bitsPerPass;
	for (int pass = 0; pass < nPasses; ++pass) {
		// Perform one pass of radix sort, sorting bitsPerPass bits 
		int lowBit = pass * bitsPerPass;
		// Set in and out vector pointers for radix sort pass 
		std::vector<MortonPrimitive>& in = (pass & 1) ? tempVector : *v;
		std::vector<MortonPrimitive>& out = (pass & 1) ? *v : tempVector;

		// Count number of zero bits in array for current radix sort bit 
		constexpr int nBuckets = 1 << bitsPerPass;
		int bucketCount[nBuckets] = { 0 };
		constexpr int bitMask = (1 << bitsPerPass) - 1;
		for (const MortonPrimitive& mp : in) {
			int bucket = (mp.mortonCode >> lowBit) & bitMask;
			++bucketCount[bucket];
		}

		// Compute starting index in output array for each bucket 
		int outIndex[nBuckets];
		outIndex[0] = 0;
		for (int i = 1; i < nBuckets; ++i)
			outIndex[i] = outIndex[i - 1] + bucketCount[i - 1];

		// Store sorted values in output array 
		for (const MortonPrimitive& mp : in) {
			int bucket = (mp.mortonCode >> lowBit) & bitMask;
			out[outIndex[bucket]++] = mp;
		}

	}
	// Copy final result from tempVector, if needed 
	if (nPasses & 1)
		std::swap(*v, tempVector);


}

BVHAccel::BVHBuildNode* BVHAccel::recursiveBuild(MemoryArena& arena,
	std::vector<BVHPrimitiveInfo>& primitiveInfo, int start,
	int end, int* totalNodes,
	std::vector<Triangle*>& orderedPrims) {
	BVHBuildNode* node = arena.Alloc<BVHBuildNode>();
	(*totalNodes)++;

	// compute bounds of all primitives in BVH node
	AABB bounds;
	for (int i = start; i < end; ++i)
		bounds = AABB::AABB::Union(bounds, primitiveInfo[i].bounds);

	int nPrimitives = end - start;
	if (nPrimitives < maxPrimsInNode) {
		// create leaf node
		int firstPrimOffset = orderedPrims.size();
		for (int i = start; i < end; ++i) {
			int primNum = primitiveInfo[i].primitiveNumber;
			orderedPrims.push_back(primitives[primNum]);
		}
		node->initLeaf(firstPrimOffset, nPrimitives, bounds);
		return node;
	}
	else {
		// Compute bound of primitive centroids, choose split dimension dim
		AABB centroidBounds; // initialize?
		for (int i = start; i < end; ++i)
			centroidBounds = AABB::AABB::Union(centroidBounds, primitiveInfo[i].centroid);
		int dim = centroidBounds.maxExtent();

		//Partition primitives into two sets and build children
		int mid = (start + end) / 2;
		if (centroidBounds.max[dim] == centroidBounds.min[dim]) {
			// Create leaf BVHBuildNode 
			int firstPrimOffset = orderedPrims.size();
			for (int i = start; i < end; ++i) {
				int primNum = primitiveInfo[i].primitiveNumber;
				orderedPrims.push_back(primitives[primNum]);
			}
			node->initLeaf(firstPrimOffset, nPrimitives, bounds);
			return node;
		}
		else {

			// Allocate BucketInfo for SAH partition buckets
			constexpr int nBuckets = 12;
			struct BucketInfo {
				int count = 0;
				AABB bounds;
			};
			BucketInfo buckets[nBuckets];

			// Initialize BucketInfo for SAH partition buckets
			for (int i = start; i < end; ++i) {
				int b = nBuckets *
					centroidBounds.Offset(primitiveInfo[i].centroid)[dim];
				if (b == nBuckets) b = nBuckets - 1;
				buckets[b].count++;
				buckets[b].bounds = AABB::AABB::Union(buckets[b].bounds, primitiveInfo[i].bounds);
			}

			// Compute costs for splitting after each bucket
			float cost[nBuckets - 1];
			for (int i = 0; i < nBuckets - 1; ++i) {
				AABB b0, b1;
				int count0 = 0, count1 = 0;
				for (int j = 0; j <= i; ++j) {
					b0 = AABB::AABB::Union(b0, buckets[j].bounds);
					count0 += buckets[j].count;
				}
				for (int j = i + 1; j < nBuckets; ++j) {
					b1 = AABB::AABB::Union(b1, buckets[j].bounds);
					count1 += buckets[j].count;
				}
				cost[i] = 0.125f + (count0 * b0.SurfaceArea() +
					count1 * b1.SurfaceArea()) / bounds.SurfaceArea();
			}

			// Find bucket to split at that minimizes SAH metric
			float minCost = cost[0];
			int minCostSplitBucket = 0;
			for (int i = 1; i < nBuckets - 1; ++i) {
				if (cost[i] < minCost) {
					minCost = cost[i];
					minCostSplitBucket = i;
				}
			}

			// Either create leaf or interior BVHBuildNode
			float leafCost = nPrimitives;
			if (nPrimitives > maxPrimsInNode || minCost < leafCost) {
				BVHPrimitiveInfo* pmid = std::partition(&primitiveInfo[start],
					&primitiveInfo[end - 1] + 1,
					[=](const BVHPrimitiveInfo& pi) {
						int b = nBuckets * centroidBounds.Offset(pi.centroid)[dim];
						if (b == nBuckets) b = nBuckets - 1;
						return b <= minCostSplitBucket;
					});
				mid = pmid - &primitiveInfo[0];
			}
			else {
				// Create leaf BVHBuildNode
				int firstPrimOffset = orderedPrims.size();
				for (int i = start; i < end; ++i) {
					int primNum = primitiveInfo[i].primitiveNumber;
					orderedPrims.push_back(primitives[primNum]);
				}
				node->initLeaf(firstPrimOffset, nPrimitives, bounds);
				return node;
			}

			if (start == mid || mid == end) {
				mid = start + (end - start) / 2;
			}

			// Partition primitives based on splitMethod
			node->InitInterior(dim,
				recursiveBuild(arena, primitiveInfo, start, mid,
					totalNodes, orderedPrims),
				recursiveBuild(arena, primitiveInfo, mid, end,
					totalNodes, orderedPrims));
			return node;
		}
	}
	return nullptr;
}

BVHAccel::BVHBuildNode* BVHAccel::buildUpperSAH(MemoryArena& arena,
	std::vector<BVHBuildNode*>& treeletRoots, int start, int end,
	int* totalNodes, int recursionDepth) const {

	int nNodes = end - start;

	if (nNodes <= 1) return treeletRoots[start];

	(*totalNodes)++;
	BVHBuildNode* node = arena.Alloc<BVHBuildNode>();

	// 计算此HLBVH结点下的所有图元边界框
	AABB bounds;
	for (int i = start; i < end; ++i)
		bounds = AABB::Union(bounds, treeletRoots[i]->bounds);

	// 计算此HLBVH结点下的所有图元质心的边界框，并选择分割轴dim
	AABB centroidBounds;
	for (int i = start; i < end; ++i) {
		glm::vec3 centroid =
			(treeletRoots[i]->bounds.min + treeletRoots[i]->bounds.max) *
			0.5f;
		centroidBounds = AABB::Union(centroidBounds, centroid);
	}
	int dim = centroidBounds.maxExtent();

	// 为HLBVH的SAH分区的桶初始化信息_BucketInfo_
	constexpr int nBuckets = 12;
	struct BucketInfo {
		int count = 0;
		AABB bounds;
	};
	BucketInfo buckets[nBuckets];
	for (int i = start; i < end; ++i) {
		float centroid = (treeletRoots[i]->bounds.min[dim] +
			treeletRoots[i]->bounds.max[dim]) *
			0.5f;
		int b =
			nBuckets * ((centroid - centroidBounds.min[dim]) /
				(centroidBounds.max[dim] - centroidBounds.min[dim]));
		if (b == nBuckets) b = nBuckets - 1;
		buckets[b].count++;
		buckets[b].bounds = AABB::Union(buckets[b].bounds, treeletRoots[i]->bounds);
	}

	// 计算每个桶的分区花费(划分点在桶位置后)
	float cost[nBuckets - 1];
	for (int i = 0; i < nBuckets - 1; ++i) {
		AABB b0, b1;
		int count0 = 0, count1 = 0;
		for (int j = 0; j <= i; ++j) {
			b0 = AABB::Union(b0, buckets[j].bounds);
			count0 += buckets[j].count;
		}
		for (int j = i + 1; j < nBuckets; ++j) {
			b1 = AABB::Union(b1, buckets[j].bounds);
			count1 += buckets[j].count;
		}
		cost[i] = .125f +
			(count0 * b0.SurfaceArea() + count1 * b1.SurfaceArea()) /
			bounds.SurfaceArea();
	}

	// 寻找最小的SAH花费
	float minCost = cost[0];
	int minCostSplitBucket = 0;
	for (int i = 1; i < nBuckets - 1; ++i) {
		if (cost[i] < minCost) {
			minCost = cost[i];
			minCostSplitBucket = i;
		}
	}

	// 在选定的SAH存储桶中创建叶子结点或拆分图元集
	BVHBuildNode** pmid = std::partition(
		&treeletRoots[start], &treeletRoots[end - 1] + 1,
		[=](const BVHBuildNode* node) {
			float centroid =
				(node->bounds.min[dim] + node->bounds.max[dim]) * 0.5f;
			int b = nBuckets *
				((centroid - centroidBounds.min[dim]) /
					(centroidBounds.max[dim] - centroidBounds.min[dim]));
			if (b == nBuckets) b = nBuckets - 1;
			return b <= minCostSplitBucket;
		});

	int mid = pmid - &treeletRoots[0];
	if (mid == start || mid == end) {
		mid = start + (end - start) / 2;
	}

	node->InitInterior(
		dim, this->buildUpperSAH(arena, treeletRoots, start, mid, totalNodes, recursionDepth),
		this->buildUpperSAH(arena, treeletRoots, mid, end, totalNodes, recursionDepth));
	return node;

}


BVHAccel::BVHBuildNode* BVHAccel::HLBVHBuild(MemoryArena& arena,
	const std::vector<BVHPrimitiveInfo>& primitiveInfo,
	int* totalNodes,
	std::vector<Triangle*>& orderedPrims) const

{
	// Compute bounding box of all primitive centroids
	AABB bounds;
	for (const BVHPrimitiveInfo& pi : primitiveInfo)
	{
		bounds = AABB::Union(bounds, pi.centroid);
	}
	//printf("bounds: %f %f %f %f %f %f\n", bounds.min.x, bounds.min.y, bounds.min.z, bounds.max.x, bounds.max.y, bounds.max.z);
	// Compute Morton indices of primitives 
	std::vector<MortonPrimitive> mortonPrims(primitiveInfo.size());
	updateMortonCodes(mortonPrims, primitiveInfo, bounds, 512);

	// apply radix sort to morton codes
	RadixSort(&mortonPrims);

	// Create LBVH treelet at bottom of the BVH
	std::vector<LBVHTreelet> treeletsToBuild;
	for (int start = 0, end = 1; end <= (int)mortonPrims.size(); ++end) {
		uint32_t mask = 0b00111111111111000000000000000000;
		if (end == (int)mortonPrims.size() ||
			((mortonPrims[start].mortonCode & mask) !=
				(mortonPrims[end].mortonCode & mask))) {
			// Add entry to treeletsToBuild for this treelet
			int nPrimitives = end - start;
			int maxBVHNodes = 2 * nPrimitives - 1;
			BVHBuildNode* nodes = arena.Alloc<BVHBuildNode>(maxBVHNodes, false);
			treeletsToBuild.push_back({ start, nPrimitives, nodes });
			start = end;
		}
	}

	std::atomic<int> atomicTotal(0), orderedPrimsOffset(0);
	orderedPrims.resize(primitives.size());
	// Create LBVHs for treelets in sequential

	for (int i = 0; i < treeletsToBuild.size(); ++i) {
		// Generate LBVH for treelet
		int nodesCreated = 0;
		const int firstBitIndex = 29 - 12; // the first 12 bits have already been used for a larger partitioning
		LBVHTreelet& tr = treeletsToBuild[i];
		tr.buildNodes =
			emitLBVH(tr.buildNodes, primitiveInfo, &mortonPrims[tr.startIndex],
				tr.nPrimitives, &nodesCreated, orderedPrims,
				&orderedPrimsOffset, firstBitIndex);
		atomicTotal += nodesCreated;
	}
	*totalNodes = atomicTotal;
	std::vector<BVHBuildNode*> finishedTreelets;
	for (LBVHTreelet& treelet : treeletsToBuild)
		finishedTreelets.push_back(treelet.buildNodes);

	return buildUpperSAH(arena, finishedTreelets, 0,
		finishedTreelets.size(), totalNodes);
}

bool __device__ BVHIntersect(const Ray& ray, LinearBVHNode* dev_nodes, Triangle* dev_triangles, ShadeableIntersection* isect) {
	bool hit = false;
	glm::vec3 invDir(1 / ray.direction.x, 1 / ray.direction.y, 1 / ray.direction.z);
	int dirIsNeg[3] = { invDir.x < 0, invDir.y < 0, invDir.z < 0 };
	// Follow ray through BVH nodes to find primitive intersections 
	int toVisitOffset = 0, currentNodeIndex = 0;
	int nodesToVisit[64];
	float tmin = FLT_MAX;
	Triangle* hitTriangle = nullptr;
	while (true) {
		LinearBVHNode node = dev_nodes[currentNodeIndex];
		// Check ray against BVH node
	
		if (node.bounds.IntersectP(ray)) {
#ifdef DEBUG_BVH
			isect->hitBVH += 0.002f;
#endif
			if (node.nPrimitives > 0) {
				// Intersect ray with primitives in leaf BVH node
				for (int i = 0; i < node.nPrimitives; ++i)
				{
#ifdef DEBUG_BVH
					isect->hitBVH += 0.002f;
#endif
					float tempt = dev_triangles[node.primitivesOffset + i].intersect(ray);
					if (tempt > 0)
						hit = true;
					if (tempt < tmin && tempt > 0)
					{
						tmin = tempt;
						hitTriangle = &dev_triangles[node.primitivesOffset + i];
					}
				}

				if (toVisitOffset == 0) break;
				currentNodeIndex = nodesToVisit[--toVisitOffset];

			}
			else {
				// Put far BVH node on nodesToVisit stack, advance to near node
				if (dirIsNeg[node.axis]) {
					nodesToVisit[toVisitOffset++] = currentNodeIndex + 1;
					currentNodeIndex = node.secondChildOffset;
				}
				else {
					nodesToVisit[toVisitOffset++] = node.secondChildOffset;
					currentNodeIndex = currentNodeIndex + 1;
				}

			}
		}
		else {
			if (toVisitOffset == 0) break;

			currentNodeIndex = nodesToVisit[--toVisitOffset];
		}

	}

	if (hit && isect)
	{
		if (tmin < isect->t || isect->t == -1.f)
		{
			isect->t = tmin;
			isect->surfaceNormal = hitTriangle->getNormal(ray.origin + ray.direction * tmin);
			isect->uv = hitTriangle->getUV(ray.origin + ray.direction * tmin);
			isect->materialId = hitTriangle->materialid;
			isect->lightId = hitTriangle->lightid;
		}
	}
	return hit;
}

// pre-order tranversal
void BVHAccel::traverseBVH(BVHBuildNode* node, int* nodeTraversed, int depth)
{
	// bfs tranversal
	std::queue<BVHBuildNode*> queue;
	queue.push(node);
	depth = 0;
	while (!queue.empty())
	{
		int size = queue.size();
		printf("depth %d, size %d\n", depth, size);
		for (int i = 0; i < size; i++)
		{
			BVHBuildNode* current = queue.front();
			printf("node %d ", current->nPrimitives);
			queue.pop();
			(*nodeTraversed)++;
			if (current->children[0])
				queue.push(current->children[0]);
			if (current->children[1])
				queue.push(current->children[1]);
		}
		printf("\n");
		depth++;
	}
}

void BVHAccel::traverseLBVH(BVHAccel::LinearBVHNode* node, int totalNodes, int depth)
{
	// bfs tranversal
	std::queue<int> q;
	q.push(0);
	depth = 0;
	while (!q.empty())
	{
		int size = q.size();
		printf("depth %d, size %d\n", depth++, size);
		for (int i = 0; i < size; ++i)
		{
			int current = q.front();
			q.pop();
			printf("%d ", node[current].nPrimitives);
			if (node[current].nPrimitives == 0)
			{
				q.push(current + 1);
				q.push(node[current].secondChildOffset);
			}
		}
		printf("\n");

	}
}

void BVHAccel::build(std::vector<Triangle>& triangles, int numTriangles) {
	if (primitives.empty())
		return;

	// calculate AABB and centroid for each primitive
	std::vector<BVHPrimitiveInfo> primitiveInfo(primitives.size());
	for (size_t i = 0; i < primitives.size(); ++i) {
		AABB bounds = primitives[i]->getBounds(); 
		primitiveInfo[i] = { static_cast<int>(i), (bounds.min + bounds.max) * 0.5f, bounds };
	}

	// construct BVH tree
	MemoryArena arena(256 * numTriangles);
	int totalNodes = 0;
	std::vector<Triangle*> orderedPrims;
	orderedPrims.reserve(primitives.size());

	BVHBuildNode* root = HLBVHBuild(arena, primitiveInfo, &totalNodes, orderedPrims);
	//BVHBuildNode* root = recursiveBuild(arena, primitiveInfo, 0, primitives.size(), &totalNodes, orderedPrims);
	// swap orderedPrims with primitives
	primitives.swap(orderedPrims);

	//traverseBVH(root, &totalNodes);

	// Create a temporary array to hold the reordered triangles
	std::vector<Triangle> tempTriangles(primitives.size());

	for (size_t i = 0; i < primitives.size(); ++i)
	{
		tempTriangles[i] = *primitives[i];
	}

	// Copy the reordered triangles back to the triangles array
	for (size_t i = 0; i < tempTriangles.size(); ++i)
	{
		triangles[i] = tempTriangles[i];
		// Update the pointer in primitives to point to the new location
		//primitives[i] = &triangles[i];
	}
	//traverseBVH(root, &totalNodes, 0);
	// linearize BVH tree
	nodes = new LinearBVHNode[totalNodes];
	int offset = 0;
	flattenBVHTree(root, &offset, totalNodes);
	//traverseLBVH(nodes, totalNodes);

	bvhNodes = totalNodes;
	// copy linearized BVH tree to device memory
	hipMalloc(&dev_nodes, totalNodes * sizeof(LinearBVHNode));
	hipMemcpy(dev_nodes, nodes, totalNodes * sizeof(LinearBVHNode), hipMemcpyHostToDevice);
	
	// check for CUDA errors
	checkCUDAError("BVHAccel::build");
}
