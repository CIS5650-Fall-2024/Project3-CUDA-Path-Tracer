#include "hip/hip_runtime.h"
#include "intersections.h"

__device__ float boxIntersectionTest(
    Geom box,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    Ray q;
    q.origin    =                multiplyMV(box.inverseTransform, glm::vec4(r.origin   , 1.0f));
    q.direction = glm::normalize(multiplyMV(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (+0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? +1 : -1;
            if (ta > 0 && ta > tmin)
            {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax)
            {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0)
    {
        outside = true;
        if (tmin <= 0)
        {
            tmin = tmax;
            tmin_n = tmax_n;
            outside = false;
        }
        intersectionPoint = multiplyMV(box.transform, glm::vec4(getPointOnRay(q, tmin), 1.0f));
        normal = glm::normalize(multiplyMV(box.invTranspose, glm::vec4(tmin_n, 0.0f)));
        return glm::length(r.origin - intersectionPoint);
    }

    return -1;
}

__device__ float sphereIntersectionTest(
    Geom sphere,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    float radius = .5;

    glm::vec3 ro = multiplyMV(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float vDotDirection = glm::dot(rt.origin, rt.direction);
    float radicand = vDotDirection * vDotDirection - (glm::dot(rt.origin, rt.origin) - powf(radius, 2));
    if (radicand < 0)
    {
        return -1;
    }

    float squareRoot = sqrt(radicand);
    float firstTerm = -vDotDirection;
    float t1 = firstTerm + squareRoot;
    float t2 = firstTerm - squareRoot;

    float t = 0;
    if (t1 < 0 && t2 < 0)
    {
        return -1;
    }
    else if (t1 > 0 && t2 > 0)
    {
        t = min(t1, t2);
        outside = true;
    }
    else
    {
        t = max(t1, t2);
        outside = false;
    }

    glm::vec3 objspaceIntersection = getPointOnRay(rt, t);

    intersectionPoint = multiplyMV(sphere.transform, glm::vec4(objspaceIntersection, 1.f));
    normal = glm::normalize(multiplyMV(sphere.invTranspose, glm::vec4(objspaceIntersection, 0.f)));
    if (!outside)
    {
        normal = -normal;
    }

    return glm::length(r.origin - intersectionPoint);
}

__device__ float intersectRayWithBoundingBox(const glm::vec3& boxMin, const glm::vec3& boxMax, const Ray& ray) {
    float tmin = -1e38f;
    float tmax = 1e38f;

    for (int i = 0; i < 3; ++i) {
        float invD = 1.0f / ray.direction[i];
        float t0 = (boxMin[i] - ray.origin[i]) * invD;
        float t1 = (boxMax[i] - ray.origin[i]) * invD;

        if (invD < 0.0f) {
            float temp = t0;
            t0 = t1;
            t1 = temp;
        }

        if (t0 > tmin) tmin = t0;
        if (t1 < tmax) tmax = t1;

        if (tmax < tmin) return -1;
    }

	return tmin;
}

__device__ float meshIntersectionTest(
    Geom geom,
    const Triangle* triangles,
    const glm::vec3* vertices,
    const glm::vec3* normals,
    const Mesh& mesh,
    int rootNodeIndex,
	const BvhNode* bvhNodes,
    const Ray& r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    bool& outside,
    int& hitTriangleIndex,
    glm::vec2& baryCoords,
    int* nodeStack) {

	// Each thread has its own stack, so read/write destination in shared memory needs to be offset.
	int offset = (threadIdx.x * MAX_BVH_DEPTH);

    // Transform the ray into object space
    Ray rt;
    rt.origin = multiplyMV(geom.inverseTransform, glm::vec4(r.origin, 1.0f));
    rt.direction = glm::normalize(multiplyMV(geom.inverseTransform, glm::vec4(r.direction, 0.0f)));
	float t = -1;
	float tMin = FLT_MAX;

	int stackIndex = 0;
	nodeStack[offset + (stackIndex++)] = rootNodeIndex; // note postfix increment
    
    // Test all triangles within the mesh
    while (stackIndex > 0) {
		BvhNode node = bvhNodes[nodeStack[offset + (--stackIndex)]]; // note prefix decrement
         
		// If the ray does not intersect the bounding box, or previous nodes have found closer intersections, skip this node
		float tBox = intersectRayWithBoundingBox(node.min, node.max, rt);
		if (tBox < 0 || tBox > tMin) {
			continue;
		}

		// If the node is a leaf node, test the triangles
        if (node.leftChild == -1 && node.rightChild == -1) {
            for (int i = node.trianglesStartIdx; i < node.trianglesStartIdx + node.numTriangles; ++i) {
                const Triangle& triangle = triangles[i];

                glm::vec3 v0 = vertices[mesh.vertStartIndex + triangle.attributeIndex[0]];
                glm::vec3 v1 = vertices[mesh.vertStartIndex + triangle.attributeIndex[1]];
                glm::vec3 v2 = vertices[mesh.vertStartIndex + triangle.attributeIndex[2]];

                glm::vec3 n0 = normals[mesh.vertStartIndex + triangle.attributeIndex[0]];
                glm::vec3 n1 = normals[mesh.vertStartIndex + triangle.attributeIndex[1]];
                glm::vec3 n2 = normals[mesh.vertStartIndex + triangle.attributeIndex[2]];

                glm::vec3 barycentricCoord;

                if (!glm::intersectRayTriangle(rt.origin, rt.direction, v0, v1, v2, barycentricCoord)) {
                    continue;
                }

                // Calculate the intersection point in world space
		        t = barycentricCoord.z;
                if (t >= tMin) continue;

                tMin = t;
                hitTriangleIndex = i;
                intersectionPoint = getPointOnRay(r, t);
                baryCoords = glm::vec2(barycentricCoord.x, barycentricCoord.y);

                // Interpolate the normal
                normal = glm::normalize(n0 * (1.0f - barycentricCoord.x - barycentricCoord.y) + n1 * barycentricCoord.x + n2 * barycentricCoord.y);

                // Transform the normal into to world space
                normal = glm::normalize(multiplyMV(geom.invTranspose, glm::vec4(normal, 0.0f)));

                // Determine if the intersection is outside
                outside = glm::dot(rt.direction, normal) < 0;
            }
        }
        else {
			nodeStack[offset + (stackIndex++)] = node.leftChild;  // note postfix increment
			nodeStack[offset + (stackIndex++)] = node.rightChild;
        }
    }

	return tMin / glm::length(rt.direction);
}
