#include "hip/hip_runtime.h"
#include "intersections.h"

__host__ __device__ float boxIntersectionTest(
    Geom box,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    Ray q;
    q.origin    =                multiplyMV(box.inverseTransform, glm::vec4(r.origin   , 1.0f));
    q.direction = glm::normalize(multiplyMV(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (+0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? +1 : -1;
            if (ta > 0 && ta > tmin)
            {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax)
            {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0)
    {
        outside = true;
        if (tmin <= 0)
        {
            tmin = tmax;
            tmin_n = tmax_n;
            outside = false;
        }
        intersectionPoint = multiplyMV(box.transform, glm::vec4(getPointOnRay(q, tmin), 1.0f));
        normal = glm::normalize(multiplyMV(box.invTranspose, glm::vec4(tmin_n, 0.0f)));
        return glm::length(r.origin - intersectionPoint);
    }

    return -1;
}

__host__ __device__ float sphereIntersectionTest(
    Geom sphere,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    float radius = .5;

    glm::vec3 ro = multiplyMV(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float vDotDirection = glm::dot(rt.origin, rt.direction);
    float radicand = vDotDirection * vDotDirection - (glm::dot(rt.origin, rt.origin) - powf(radius, 2));
    if (radicand < 0)
    {
        return -1;
    }

    float squareRoot = sqrt(radicand);
    float firstTerm = -vDotDirection;
    float t1 = firstTerm + squareRoot;
    float t2 = firstTerm - squareRoot;

    float t = 0;
    if (t1 < 0 && t2 < 0)
    {
        return -1;
    }
    else if (t1 > 0 && t2 > 0)
    {
        t = min(t1, t2);
        outside = true;
    }
    else
    {
        t = max(t1, t2);
        outside = false;
    }

    glm::vec3 objspaceIntersection = getPointOnRay(rt, t);

    intersectionPoint = multiplyMV(sphere.transform, glm::vec4(objspaceIntersection, 1.f));
    normal = glm::normalize(multiplyMV(sphere.invTranspose, glm::vec4(objspaceIntersection, 0.f)));
    if (!outside)
    {
        normal = -normal;
    }

    return glm::length(r.origin - intersectionPoint);
}

__host__ __device__ float meshIntersectionTestNaive(
    Geom mesh,
    Ray r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    bool& outside) {
    
    float t = INFINITY;
    glm::vec3 finalIntersectionPoint;
    glm::vec3 finalNormal;
    bool finalOutside;

    glm::vec3 originLocal = multiplyMV(mesh.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 directionLocal = glm::normalize(multiplyMV(mesh.inverseTransform, glm::vec4(r.direction, 0.0f)));

    for (int i = 0; i < mesh.numTriangles; i++) {
        const Triangle &tri = mesh.devTriangles[i];
        glm::vec3 localBarycentricCoords;

        bool hit = glm::intersectLineTriangle(
            originLocal,                             // Ray origin
            directionLocal,                          // Ray direction
            tri.points[0], tri.points[1], tri.points[2],  // Triangle vertices
            localBarycentricCoords);  

        if (!hit) {
            continue;
        }

        glm::vec3 intersectionPointLocal = localBarycentricCoords.x * tri.points[0] +
                           localBarycentricCoords.y * tri.points[1] +
                           localBarycentricCoords.z * tri.points[2];

        glm::vec3 intersectionPointWorld = multiplyMV(mesh.transform, glm::vec4(intersectionPointLocal, 1.0f));
        float currentT = glm::distance(r.origin, intersectionPointWorld) / glm::length(r.direction);

        if (currentT > t) {
            continue;
        }

        t = currentT;
        finalIntersectionPoint = intersectionPointWorld;
        glm::vec3 normalLocal = localBarycentricCoords.x * tri.normals[0] +
                                localBarycentricCoords.y * tri.normals[1] +
                                localBarycentricCoords.z * tri.normals[2];
        finalNormal = glm::normalize(multiplyMV(mesh.invTranspose, glm::vec4(normalLocal, 0.0f)));
        finalOutside = glm::dot(finalNormal, r.direction) < 0;
    }

    if (t == INFINITY) {
        return -1;
    }

    intersectionPoint = finalIntersectionPoint;
    normal = finalNormal;
    outside = finalOutside;
    return t;
}