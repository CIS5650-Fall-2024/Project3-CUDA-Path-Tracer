#include "hip/hip_runtime.h"
#include "intersections.h"

__host__ __device__ float boxIntersectionTest(
    Geom box,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    Ray q;
    q.origin    =                multiplyMV(box.inverseTransform, glm::vec4(r.origin   , 1.0f));
    q.direction = glm::normalize(multiplyMV(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (+0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? +1 : -1;
            if (ta > 0 && ta > tmin)
            {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax)
            {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0)
    {
        outside = true;
        if (tmin <= 0)
        {
            tmin = tmax;
            tmin_n = tmax_n;
            outside = false;
        }
        intersectionPoint = multiplyMV(box.transform, glm::vec4(getPointOnRay(q, tmin), 1.0f));
        normal = glm::normalize(multiplyMV(box.invTranspose, glm::vec4(tmin_n, 0.0f)));
        return glm::length(r.origin - intersectionPoint);
    }

    return -1;
}

__host__ __device__ float sphereIntersectionTest(
    Geom sphere,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    float radius = .5;

    glm::vec3 ro = multiplyMV(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float vDotDirection = glm::dot(rt.origin, rt.direction);
    float radicand = vDotDirection * vDotDirection - (glm::dot(rt.origin, rt.origin) - powf(radius, 2));
    if (radicand < 0)
    {
        return -1;
    }

    float squareRoot = sqrt(radicand);
    float firstTerm = -vDotDirection;
    float t1 = firstTerm + squareRoot;
    float t2 = firstTerm - squareRoot;

    float t = 0;
    if (t1 < 0 && t2 < 0)
    {
        return -1;
    }
    else if (t1 > 0 && t2 > 0)
    {
        t = min(t1, t2);
        outside = true;
    }
    else
    {
        t = max(t1, t2);
        outside = false;
    }

    glm::vec3 objspaceIntersection = getPointOnRay(rt, t);

    intersectionPoint = multiplyMV(sphere.transform, glm::vec4(objspaceIntersection, 1.f));
    normal = glm::normalize(multiplyMV(sphere.invTranspose, glm::vec4(objspaceIntersection, 0.f)));

    return glm::length(r.origin - intersectionPoint);
}

bool rayIntersectsAABB(const Ray& ray, const glm::vec3& min, const glm::vec3& max) {
	// Slabs Method for Ray-AABB intersection
	float tmin = (min.x - ray.origin.x) / ray.direction.x;
	float tmax = (max.x - ray.origin.x) / ray.direction.x;

	if (tmin > tmax) std::swap(tmin, tmax);

	float tymin = (min.y - ray.origin.y) / ray.direction.y;
	float tymax = (max.y - ray.origin.y) / ray.direction.y;

	if (tymin > tymax) std::swap(tymin, tymax);

	if ((tmin > tymax) || (tymin > tmax))
		return false;

	if (tymin > tmin)
		tmin = tymin;

	if (tymax < tmax)
		tmax = tymax;

	float tzmin = (min.z - ray.origin.z) / ray.direction.z;
	float tzmax = (max.z - ray.origin.z) / ray.direction.z;

	if (tzmin > tzmax) std::swap(tzmin, tzmax);

	if ((tmin > tzmax) || (tzmin > tmax))
		return false;

	return true;
}

__host__ __device__ 
float meshIntersectionTest(
    Geom mesh,
	Triangle* tris,
    Ray ray,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
	glm::vec2& uvOut,
    bool& outside) {
#if BOUNDING_VOLUME_INTERSECTION_CULLING_ENABLED
	// culling box test
    if (!rayIntersectsAABB(ray, mesh.min, mesh.max)){
        return -1.0f;
    }
#endif
	glm::vec3 objOrigin = multiplyMV(mesh.inverseTransform, glm::vec4(ray.origin, 1.0f));
	glm::vec3 objDir = glm::normalize(multiplyMV(mesh.inverseTransform, glm::vec4(ray.direction, 0.0f)));

	float closestT = FLT_MAX;
	glm::vec3 closestNormal(0.0f);
	glm::vec2 closestUV(0.0f);
	bool hit = false;

	for (int i = mesh.startTriangleIndex; i <= mesh.endTriangleIndex; ++i) {
		const Triangle& tri = tris[i];
		glm::vec3 baryPosition;

		if (glm::intersectRayTriangle(objOrigin, objDir,
			tri.v0.position, tri.v1.position, tri.v2.position,
			baryPosition)) {
			float t = baryPosition.z;
			if (t > 0.0f && t < closestT) {
				closestT = t;
				hit = true;

                if (mesh.hasNormals) {
                    glm::vec3 n0 = tri.v0.normal;
                    glm::vec3 n1 = tri.v1.normal;
                    glm::vec3 n2 = tri.v2.normal;
                    closestNormal = glm::normalize(
                        (1.0f - baryPosition.x - baryPosition.y) * n0 +
                        baryPosition.x * n1 +
                        baryPosition.y * n2
                    );

				}else {
					closestNormal = glm::cross(tri.v1.position - tri.v0.position, tri.v2.position - tri.v0.position);
				}

                if (mesh.hasUVs) {
					glm::vec2 uv0 = tri.v0.uv;
					glm::vec2 uv1 = tri.v1.uv;
					glm::vec2 uv2 = tri.v2.uv;
					closestUV = (1.0f - baryPosition.x - baryPosition.y) * uv0 +
						baryPosition.x * uv1 +
						baryPosition.y * uv2;
                }else {
					closestUV = glm::vec2(0.0f);
                }
			}
		}
	}

	if (!hit) {
		return -1.0f;
	}

	glm::vec3 objIntersect = objOrigin + closestT * objDir;
	intersectionPoint = multiplyMV(mesh.transform, glm::vec4(objIntersect, 1.0f));
	normal = glm::normalize(multiplyMV(mesh.invTranspose, glm::vec4(closestNormal, 0.0f)));
	uvOut = closestUV;
	return glm::length(ray.origin - intersectionPoint);
}

