#include "hip/hip_runtime.h"
#include "interactions.h"

__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine &rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng)); // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}

__host__ __device__ float fresnelDielectric(float cosThetaI, float IOR) {
    float etaI = 1.f;
    float etaT = IOR;
    etaT = etaT < EPSILON ? 1.55f : etaT;
    cosThetaI = glm::clamp(cosThetaI, -1.f, 1.f);

    if (cosThetaI > 0.f) {
        float temp = etaI;
        etaI = etaT;
        etaT = temp;
    }
    cosThetaI = glm::abs(cosThetaI);

    // Computer cosThetaT using Snell's law
    float sinThetaI = glm::sqrt(glm::max(0.f, 1.f - cosThetaI * cosThetaI));
    float sinThetaT = etaI / etaT * sinThetaI;

    // Handle total internal reflection
    if (sinThetaT >= 1.0f) {
        return 1.f;
    }

    // Compute Fresnel reflectance using light polarization eqns, see PBRT 8.2.1
    float cosThetaT = glm::sqrt(glm::max(0.f,
        1.f - sinThetaT * sinThetaT));
    float Rparl = ((etaT * cosThetaI) - (etaI * cosThetaT)) /
        ((etaT * cosThetaI) + (etaI * cosThetaT));
    float Rperp = ((etaI * cosThetaI) - (etaT * cosThetaT)) /
        ((etaI * cosThetaI) + (etaT * cosThetaT));

    return (Rparl * Rparl + Rperp * Rperp) * 0.5f; // coefficient
}

__host__ __device__ glm::vec3 sample_f_specular_reflection(
    glm::vec3 normal, glm::vec3 rayDir, glm::vec3 color, glm::vec3& wiW) {
    wiW = glm::reflect(rayDir, normal);
    return color;
}

__host__ __device__ glm::vec3 sample_f_specular_transmission(
	glm::vec3 &normal, glm::vec3 rayDir, float IOR, glm::vec3 color, glm::vec3 &wiW) {

    // Determine if we're entering or exiting the material
    bool entering = glm::dot(rayDir, normal) < 0.0f;
    float eta = entering ? 1.0f / IOR : IOR;

    normal = entering ? normal : -normal;
    wiW = glm::refract(rayDir, normal, eta);
    
	// Total internal reflection
    if (glm::length(wiW) < EPSILON) {
        return glm::vec3(0.0f);
    }
    return color;
}

__host__ __device__ glm::vec3 sample_f_glass(
    glm::vec3& normal, glm::vec3 rayDir, float IOR, glm::vec3 color,
    glm::vec3& wiW, thrust::default_random_engine& rng) {
    thrust::uniform_real_distribution<float> u01(0, 1);
    float random = u01(rng);
	float fresnel = fresnelDielectric(glm::dot(normal, rayDir), IOR);
    if (random < 0.5f) {
        // Reflection
		wiW = glm::reflect(rayDir, normal);
		return fresnel * color * 2.f;
    }
    else {
        // Refraction
        glm::vec3 T = sample_f_specular_transmission(normal, rayDir, IOR, color, wiW);
        return 2.0f * T * (1.0f - fresnel);
    }
}

__host__ __device__ glm::vec3 sample_f_specular_plastic(
    glm::vec3 normal, glm::vec3 rayDir, glm::vec3 color, float roughness,
    glm::vec3 &wiW, thrust::default_random_engine& rng, float& pdf) {
    thrust::uniform_real_distribution<float> u01(0, 1);
    float random = u01(rng);
    if (random < (1.f - roughness)) {
        // Reflection
        wiW = glm::reflect(rayDir, normal);
		pdf = 1.0f;
        return color / glm::abs(glm::dot(wiW, normal));
    }
    else {
        // Diffuse
        glm::vec3 T = color / PI;
        wiW = calculateRandomDirectionInHemisphere(normal, rng);
        pdf = glm::cos(glm::acos(glm::dot(wiW, normal))) / PI;
        return roughness > 0 ? 1 / roughness * T : T;
    }
}

__host__ __device__ float presence_single(glm::vec3 a, glm::vec3 b) {
    return glm::dot(a, b) / glm::dot(a, glm::vec3(1.));
}

__host__ __device__ float presence(glm::vec3 a, glm::vec3 b) {
     float t = presence_single(a, b);
     t = pow(t, 3.);
     return glm::clamp(t, 0.f, 1.f);
}

__host__ __device__ void scatterRay(
    PathSegment & pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    const Material &m,
    thrust::default_random_engine &rng,
    glm::vec3 texture_color)
{
    glm::vec3 wiW;
    glm::vec3 bsdf;
	float pdf;
	bool ignore_pdf = true;

#if DEBUG_ALBEDO
    pathSegment.color = texture_color[0] == -1.0f ? m.color: texture_color;
    pathSegment.remainingBounces = 0;
    return;
#endif

    // A basic implementation of pure-diffuse shading will just call the
    // calculateRandomDirectionInHemisphere defined above.
    if (m.hasReflective && m.hasRefractive) {
		// Transparent and reflective material like glass
#if DISPERSION
        float consumeChance = 1 - presence(pathSegment.color, m.color);
        thrust::uniform_real_distribution<float> u01(0.f, 0.1f);
        if (u01(rng) < consumeChance) {
            return;
        }
        bsdf = sample_f_glass(normal, pathSegment.ray.direction, 
            //m.indexOfRefraction,
            m.indexOfRefraction + m.dispersion * 1e5 / (pathSegment.waveLength * pathSegment.waveLength), 
            m.color,
            //pathSegment.waveColor, 
            wiW, rng);
#else
            bsdf = sample_f_glass(normal, pathSegment.ray.direction, m.indexOfRefraction, m.color, wiW, rng);
#endif
		/*pdf = 1.0f;
		ignore_pdf = false;*/
    }
    else if (m.hasPlastic) {
        // Reflective material that has color, like smooth plastic
		glm::vec3 color = texture_color[0] == -1.0f ? m.color : texture_color;
        bsdf = sample_f_specular_plastic(normal, glm::normalize(pathSegment.ray.direction), color, m.roughness, wiW, rng, pdf);
		ignore_pdf = false;
    }
	else if (m.hasReflective) {
        // Acts like a mirror, no diffuse component
		bsdf = sample_f_specular_reflection(normal, pathSegment.ray.direction, m.color, wiW);
	}
	else if (m.hasRefractive) {
		// Transparent material that only transmits
		bsdf = sample_f_specular_transmission(normal, pathSegment.ray.direction, m.indexOfRefraction, m.color, wiW);
		//bsdf = glm::vec3(1.0f, 0.f, 0.f);
	}
	else {
		bsdf = texture_color[0] == -1.0f ? m.color / PI : texture_color / PI;
		wiW = calculateRandomDirectionInHemisphere(normal, rng);
		pdf = glm::cos(glm::acos(glm::dot(wiW, normal))) / PI;
		ignore_pdf = false;
	}

	pathSegment.color *= ignore_pdf ? bsdf : bsdf * glm::abs(glm::dot(wiW, normal)) / pdf;
	pathSegment.ray.direction = glm::normalize(wiW);
	pathSegment.ray.origin = intersect + pathSegment.ray.direction * 0.001f;
	pathSegment.remainingBounces--;
}
