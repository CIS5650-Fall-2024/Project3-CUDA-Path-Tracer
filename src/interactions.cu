#include "hip/hip_runtime.h"
#include "interactions.h"

__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine &rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng)); // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}

__host__ __device__ void scatterRay(
    PathSegment & pathSegment, 
    glm::vec3 intersect,
    glm::vec3 normal,
    const Material &m,
    thrust::default_random_engine &rng)
{
    // Pre-fetch
    const glm::vec3 direction = glm::normalize(pathSegment.ray.direction);
    glm::vec3 norm = glm::normalize(normal);

    pathSegment.ray.origin = intersect;
    const glm::vec3 delta = 0.001f * norm;
    pathSegment.color *= m.color;
    --pathSegment.remainingBounces;

    // Diffuse for any material
    thrust::uniform_real_distribution<float> u01(0, 1);
    float r01;

    // Specular reflection
    if (m.hasReflective > 0)
    {
        if (m.roughness <= EPSILON)
        {
            pathSegment.ray.direction = glm::reflect(direction, norm);
        }
        else
        {
            r01 = u01(rng);
            glm::vec3 randomDirectionDelta{ glm::normalize(calculateRandomDirectionInHemisphere(norm, rng)) };
            pathSegment.ray.direction = (1 - m.roughness) * glm::normalize(glm::reflect(direction, norm))
                + m.roughness * randomDirectionDelta;
        }
        pathSegment.ray.origin += delta;
    }
    // Refractive
    else if (m.hasRefractive > 0)
    {
        r01 = u01(rng);

        // Derive reflection coeff R_theta
        const float cos_theta = -glm::dot(norm, direction);
        constexpr float n_i = 1.0f;
        const float n_o = m.indexOfRefraction;
        const float R_0 = glm::pow((n_i - n_o) / (n_i + n_o), 2.0f);
        const float R_theta = R_0 + (1.0f - R_0) * glm::pow(1.0f - cos_theta, 5.0f);
        float dot_n_d = glm::dot(norm, direction);

        if (r01 > R_theta)
        {
            // Refract
            float ratio;
            if (dot_n_d > 0.f)  // material -> air
            {
                norm = -norm;
                ratio = m.indexOfRefraction;
            }
            else  // air -> material
            {
                ratio = 1.0f / m.indexOfRefraction;
            }
            pathSegment.ray.direction = glm::refract(direction, norm, ratio);

            pathSegment.ray.origin -= delta;
        }
        else
        {
            // Reflect
            pathSegment.ray.direction = glm::reflect(direction, norm);
            pathSegment.ray.origin += delta;
        }

        if (m.roughness > EPSILON)
        {
            if ((r01 > R_theta && dot_n_d < 0.f) || (r01 < R_theta && dot_n_d > 0.f)) // BRDF and BTDF
            {
                r01 = u01(rng);
                glm::vec3 randomDirectionDelta{ glm::normalize(calculateRandomDirectionInHemisphere(norm, rng)) };
                pathSegment.ray.direction = (1 - m.roughness) * pathSegment.ray.direction + m.roughness * randomDirectionDelta;
            }
        }
    }
    // Diffuse material
    else
    {
        pathSegment.ray.direction = calculateRandomDirectionInHemisphere(norm, rng);
        pathSegment.ray.origin += delta;
    }
}
