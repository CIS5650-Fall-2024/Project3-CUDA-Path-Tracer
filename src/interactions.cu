#include "hip/hip_runtime.h"
#include <>
#include <thrust/swap.h>

#include "interactions.h"

__device__ glm::vec3 squareToHemisphereCosine(glm::vec2 xi) {
  float x = xi.x;
  float y = xi.y;
  if (x == 0 && y == 0)
    return glm::vec3(0, 0, 0);

  float phi = 0.f;
  float radius = 1.f;
  float a = (2.f * x) - 1.f;
  float b = (2.f * y) - 1.f;

  // Uses squares instead of absolute values
  if ((a * a) > (b * b)) {
    // Top half
    radius *= a;
    phi = (PI / 4) * (b / a);
  }
  else {
    // Bottom half
    radius *= b;
    phi = (PI / 2) - ((PI / 4) * (a / b));
  }

  // Map the distorted Polar coordinates (phi,radius)
  // into the Cartesian (x,y) space
  glm::vec3 disc(0.f, 0.f, 0.f);
  disc.x = glm::cos(phi) * radius;
  disc.y = glm::sin(phi) * radius;

  // I think this ensures this is a hemisphere and not a sphere ? 
  disc.z = glm::sqrt(1.f - (disc.x * disc.x) - (disc.y * disc.y));

  return disc;
}

__device__ glm::vec3 localToWorld(const glm::vec3& normal, const glm::vec3& vec) {
  glm::vec3 tangent, bitangent;

  // create coordinate system from normal 
  if (glm::abs(normal.x) > glm::abs(normal.y)) {
    tangent = glm::vec3(-normal.z, 0, normal.x) / glm::sqrt(normal.x * normal.x + normal.z * normal.z);
  }
  else {
    tangent = glm::vec3(0, normal.z, -normal.y) / glm::sqrt(normal.y * normal.y + normal.z * normal.z);
  }
  bitangent = glm::cross(normal, tangent);
  return glm::mat3(glm::normalize(tangent), glm::normalize(bitangent), glm::normalize(normal)) * vec;
}

__device__ glm::vec3 worldToLocal(const glm::vec3& normal, const glm::vec3& vec) {
  glm::vec3 tangent, bitangent;
  // create coordinate system from normal 
  if (glm::abs(normal.x) > glm::abs(normal.y)) {
    tangent = glm::vec3(-normal.z, 0, normal.x) / glm::sqrt(normal.x * normal.x + normal.z * normal.z);
  }
  else {
    tangent = glm::vec3(0, normal.z, -normal.y) / glm::sqrt(normal.y * normal.y + normal.z * normal.z);
  }
  bitangent = glm::cross(normal, tangent);
  return glm::transpose(glm::mat3(glm::normalize(tangent), glm::normalize(bitangent), glm::normalize(normal))) * vec;
}

// Fresnel Dielectric adapted from PBRTv3 source code
// Copyright(c) 1998-2016 Matt Pharr, Greg Humphreys, and Wenzel Jakob.
__device__ float fresnelDielectric(float cosThetaI, float etaI, float etaT) {
  cosThetaI = glm::clamp(cosThetaI, -1.f, 1.f);
  // Potentially swap indices of refraction
  bool entering = cosThetaI > 0.f;
  if (!entering) {
    thrust::swap(etaI, etaT);
    cosThetaI = abs(cosThetaI);
  }

  // Compute _cosThetaT_ using Snell's law
  float sinThetaI = sqrtf(fmaxf(0.f, 1.f - cosThetaI * cosThetaI));
  float sinThetaT = etaI / etaT * sinThetaI;

  // Handle total internal reflection
  if (sinThetaT >= 1) return 1;
  float cosThetaT = sqrtf(fmaxf(0.f, 1.f - sinThetaT * sinThetaT));
  float Rparl = ((etaT * cosThetaI) - (etaI * cosThetaT)) /
    ((etaT * cosThetaI) + (etaI * cosThetaT));
  float Rperp = ((etaI * cosThetaI) - (etaT * cosThetaT)) /
    ((etaI * cosThetaI) + (etaT * cosThetaT));
  return (Rparl * Rparl + Rperp * Rperp) * 0.5f;
}

// Refract adapted from PBRTv3 source code
// Copyright(c) 1998-2016 Matt Pharr, Greg Humphreys, and Wenzel Jakob.
__device__ bool Refract(const glm::vec3& wi, const glm::vec3& n, float eta, glm::vec3& wt) {
  // Compute $\cos \theta_\roman{t}$ using Snell's law
  float cosThetaI = glm::dot(n, wi);
  float sin2ThetaI = fmaxf(0.f, float(1.f - cosThetaI * cosThetaI));
  float sin2ThetaT = eta * eta * sin2ThetaI;

  // Handle total internal reflection for transmission
  if (sin2ThetaT >= 1.f) return false;
  float cosThetaT = sqrtf(1 - sin2ThetaT);
  wt = eta * -wi + (eta * cosThetaI - cosThetaT) * glm::vec3(n);
  return true;
}

__device__ void scatterRay(
    PathSegment & pathSegment,
    ShadeableIntersection& intersection,
    const Material &m,
    const hipTextureObject_t* textures,
    thrust::default_random_engine &rng)
{
  thrust::uniform_real_distribution<float> u01(0, 1);
  glm::vec2 xi(u01(rng), u01(rng));
  float pdf = 0.0f; 
  glm::vec3 materialColor = m.color; 
  glm::vec3 normal = intersection.surfaceNormal; 

  if (m.textureIdx.albedo != -1) {
    float4 texCol = tex2D<float4>(textures[m.textureIdx.albedo], intersection.texSample.s, intersection.texSample.t);
    materialColor = glm::vec3(texCol.x, texCol.y, texCol.z) * m.color;
  }

  if (m.textureIdx.normal != -1) {
    float4 texNorCol = tex2D<float4>(textures[m.textureIdx.normal], intersection.texSample.s, intersection.texSample.t);
    normal = glm::vec3(texNorCol.x, texNorCol.y, texNorCol.z);
    normal = (normal * 2.f) - 1.f;
    normal = glm::normalize(localToWorld(intersection.surfaceNormal, normal));
  }

  if (m.emittance > 0.0f) {                             // EMISSION
    pathSegment.color *= (m.color * m.emittance);
    pathSegment.isFinished = true;
  }
  else if (m.hasReflective > 0.f) {                     // PERFECT SPECULAR REFLECTION
    glm::vec3 woW = glm::normalize(pathSegment.ray.direction);
    glm::vec3 wo = worldToLocal(normal, woW);  // there is no need for going to a local coord space, but it's for my own learning benefit

    // perfect specular direction
    glm::vec3 wi = glm::normalize(glm::reflect(wo, glm::vec3(0.f, 0.f, 1.f)));

    pathSegment.color *= materialColor;
    pathSegment.color = glm::clamp(pathSegment.color, 0.f, 1.f);

    wi = glm::normalize(localToWorld(normal, wi)); 

    // new ray for the next bounce
    pathSegment.ray.origin = pathSegment.ray.origin + (intersection.t * pathSegment.ray.direction);
    pathSegment.ray.origin += EPSILON * wi;   // slightly offset the ray origin in the direction of the ray direction
    pathSegment.ray.direction = wi;

    --pathSegment.remainingBounces;
  }
  else if (m.hasTransmissive > 0.f) {                   // FRESNEL SPECULAR (Glass) 
    glm::vec3 wo = worldToLocal(normal, -pathSegment.ray.direction); 
    glm::vec3 wi; 
    glm::vec3 bsdfValue; 

    float etaA = 1.f; 
    float etaB = 1.5f; 

    float fresnelTerm = fresnelDielectric(wo.z, etaA, etaB); 
    if (xi.x < fresnelTerm) {
      // reflection
      wi = glm::vec3(-wo.x, -wo.y, wo.z); 
      pdf = fresnelTerm; 
      bsdfValue = fresnelTerm * materialColor / abs(wi.z); 
    }
    else 
    {
      // transmission
      bool entering = wo.z > 0;
      float etaI = entering ? etaA : etaB;
      float etaT = entering ? etaB : etaA;

      glm::vec3 normalLocal = glm::vec3(0, 0, 1);

      // ensure normal is on the same side of the incident vector
      normalLocal = glm::dot(normalLocal, wo) < 0.f ? -normalLocal : normalLocal;

      if (!Refract(wo, normalLocal, etaI / etaT, wi)) {
        pathSegment.isTerminated = true;
        return;
      }
      glm::vec3 ft = materialColor * (1.f - fresnelTerm); 

      pdf = 1.f - fresnelTerm; 
      bsdfValue = ft / abs(wi.z); 
    }

    wi = glm::normalize(localToWorld(normal, wi));

    pathSegment.color *= bsdfValue * glm::abs(glm::dot(wi, normal)) / pdf;
    pathSegment.color = glm::clamp(pathSegment.color, 0.f, 1.f);

    // new ray for the next bounce
    pathSegment.ray.origin = pathSegment.ray.origin + (intersection.t * pathSegment.ray.direction);
    pathSegment.ray.origin += 1.f * wi;   // slightly offset the ray origin in the direction of the ray direction
    pathSegment.ray.direction = wi;

    --pathSegment.remainingBounces; 
  }
  else {                                                // PERFECT DIFFUSE REFLECTION
    // generate random direction in hemisphere
    glm::vec3 wi = squareToHemisphereCosine(xi);

    // get the pdf (square to hemisphere cosine)
    pdf = glm::abs(wi.z) * INV_PI;

    if (pdf < EPSILON || isnan(pdf)) {
      pathSegment.isTerminated = true;
      return;
    }

    glm::vec3 bsdfValue = materialColor * INV_PI;

    // convert vec3 into the world coordinate system (using surface normal)
    wi = glm::normalize(localToWorld(intersection.surfaceNormal, wi));

    // update throughput
    pathSegment.color *= bsdfValue * glm::abs(glm::dot(wi, normal)) / pdf;
    pathSegment.color = glm::clamp(pathSegment.color, 0.f, 1.f);

    // new ray for the next bounce
    pathSegment.ray.origin = pathSegment.ray.origin + (intersection.t * pathSegment.ray.direction);
    pathSegment.ray.origin += EPSILON * wi;   // slightly offset the ray origin in the direction of the ray direction
    pathSegment.ray.direction = wi;

    --pathSegment.remainingBounces;
  }
}
