#include "hip/hip_runtime.h"
#include "interactions.h"

__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine &rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng)); // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}

__host__ __device__ void scatter_ray(
    PathSegment &path_segment,
    const ShadeableIntersection &intersection,
    const Material &m,
    thrust::default_random_engine &rng)
{
    // TODO: implement this.
    // A basic implementation of pure-diffuse shading will just call the
    // calculateRandomDirectionInHemisphere defined above.

    const auto intersection_point = path_segment.ray.origin + path_segment.ray.direction * (intersection.t - EPSILON);

    thrust::uniform_real_distribution<float> u01{0, 1};
    // const auto probability = u01(rng);
    const auto probability = 1.0f;

    if (probability < m.hasReflective) {
        path_segment.ray.direction = glm::normalize(glm::reflect(path_segment.ray.direction, intersection.surfaceNormal));
        path_segment.ray.origin = intersection_point;
        path_segment.color *= m.specular.color;
    } else {
        path_segment.ray.direction = glm::normalize(calculateRandomDirectionInHemisphere(intersection.surfaceNormal, rng));
        path_segment.ray.origin = intersection_point;
    }

    path_segment.color *= m.color;
}
