#include "hip/hip_runtime.h"
﻿#include "interactions.h"

__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine &rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

	// The random generated direction is cosine weighted by sqrt the random number
    float up = sqrt(u01(rng)); // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

	// the final direction is a combination of a linear combination of the two perpendicular directions and the normal
    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}

__device__ glm::vec3 getEnvironmentalRadiance(glm::vec3 direction, hipTextureObject_t envMap) {
    float theta = acosf(direction.y);         // θ
    float phi = atan2f(direction.z, direction.x); // φ
    if (phi < 0) phi += 2.0f * PI;

    float u = phi / (2.0f * PI);            // [0, 1]
    float v = theta / PI;                   // [0, 1]
	if (envMap == NULL) return glm::vec3(0.0f); // return black if no envMap (for debugging purposes
	float4 texel = tex2D<float4>(envMap, u, v);
	return glm::vec3(texel.x, texel.y, texel.z);
}

__device__ void scatterRay(
    PathSegment & pathSegment,
    glm::vec3 intersect,
    float t,
    glm::vec3 normal, 
	glm::vec2 uv,
    const Material &m,
    thrust::default_random_engine &rng)
{

	glm::vec3 wi = glm::vec3(0.0f);
	glm::vec3 col = glm::vec3(0.0f);

    // TODO: implement PBR model
 //   glm::vec3 L = slerp(glm::reflect(pathSegment.ray.direction, normal), calculateRandomDirectionInHemisphere(normal, rng), m.roughness);
	//L = glm::normalize(L);
	//glm::vec3 H = glm::normalize(L + pathSegment.ray.direction);
	//float NdotL = glm::dot(normal, L);
	//float NdotV = glm::dot(normal, -pathSegment.ray.direction);
	//float NdotH = glm::dot(normal, H);
    //glm::vec3 F = fresnelSchlick(NdotL, glm::vec3(m.metallic));
    //glm::vec3 kd = (1.0f - F) * m.color / PI;

    if (m.reflective == 1.0f)
    {
		// perfect reflection
		wi = glm::reflect(pathSegment.ray.direction, normal);
		col = m.color;
	}
    else
    {
        // Ideal diffuse
		wi = calculateRandomDirectionInHemisphere(normal, rng);
        col = m.color;
    }

    pathSegment.remainingBounces--;

#ifdef DEBUG_NORMAL
    col = glm::vec3(1.f);
    pathSegment.color = DEBUG_NORMAL ? (normal + 1.0f) / 2.0f : normal;
	pathSegment.remainingBounces = 0;
#elif defined(DEBUG_WORLD_POS)
	col = glm::vec3(1.f);
    pathSegment.color = glm::clamp(intersect, glm::vec3(0), glm::vec3(1.0f));
	pathSegment.remainingBounces = 0;
#elif defined(DEBUG_UV)
	col = glm::vec3(1.f);
	pathSegment.color = glm::vec3(uv, 0);
	pathSegment.remainingBounces = 0;
#endif

	pathSegment.ray.origin = intersect;
    pathSegment.ray.direction = glm::normalize(wi);
    pathSegment.throughput *= col;
}
