#include "hip/hip_runtime.h"
#include "interactions.h"

__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine &rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng)); // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}

__host__ __device__ void sampleDiffuse(
    PathSegment& pathSegment,
    glm::vec3 normal,
    const Material& m,
    float prob)
{
   pathSegment.ray.direction = calculateRandomDirectionInHemisphere(normal, rng);
   pathSegment.color *= m.color / prob;
}

__host__ __device__ void sampleRefl(
    PathSegment& pathSegment,
    glm::vec3 normal,
    const Material& m,
    float prob)
{
    pathSegment.ray.direction = glm::reflect(pathSegment.ray.direction, normal);
    pathSegment.color *= m.specular.color / prob;
}

__host__ __device__ void sampleRefract(
    PathSegment& pathSegment,
    glm::vec3 normal,
    const Material& m,
    float prob)
{
    float etaA = 1.f;
    float etaB = m.indexOfRefraction;
    bool entering = (glm::dot(-pathSegment.ray.direction, normal) > 0);
    float etaI = entering ? etaA : etaB;
    float etaT = entering ? etaB : etaA;

    glm::vec3 N = entering ? normal : -normal;

    glm::vec3 refractedDir = glm::refract(pathSegment.ray.direction, N, etaI / etaT);

    if (glm::length(refractedDir) > 0.f) {
        pathSegment.ray.direction = refractedDir;
    }
    else {
        //total internal reflection
        pathSegment.ray.direction = glm::reflect(pathSegment.ray.direction, N);
    }
    pathSegment.color *= m.specular.color / prob;
}

__host__ __device__ void scatterRay(
    PathSegment & pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    const Material &m,
    thrust::default_random_engine &rng)
{
    float probDiffuse = 0.f;

    pathSegment.remainingBounces--;

    float totalIntensity = glm::length(m.color) + glm::length(m.specular.color);

    if (totalIntensity > 0.f) {
        probDiffuse = glm::length(m.color) / totalIntensity;
    }

    thrust::uniform_real_distribution<float> u01(0, 1);
    float rand = u01(rng);

    if (rand < probDiffuse) {
        //diffuse shading
        sampleDiffuse(pathSegment, normal, m, probDiffuse);
    }
    else {
        if (m.hasReflective > 0.f && m.hasRefractive > 0.f) {
            float denom = m.hasReflective + m.hasRefractive;
            float probReflect = (1.f - probDiffuse) * m.hasReflective / denom;
            float probRefract = (1.f - probDiffuse) * m.hasRefractive / denom;
            if (rand < probDiffuse + probReflect) {
                //reflection
                //divide color by probReflect
                sampleRefl(pathSegment, normal, m, probReflect);
            }
            else {
                //refraction
                //divide color by probRefract
                sampleRefract(pathSegment, normal, m, probRefract);
            }
        } 
        else if (m.hasReflective > 0.f) {
            //reflection
            //divide color by 1 - probDiffuse
            sampleRefl(pathSegment, normal, m, 1.f - probDiffuse);
        }
        else if (m.hasRefractive > 0.f) {
            //refraction
            //divide color by 1 - probDiffsue
            sampleRefract(pathSegment, normal, m, 1.f - probDiffuse);
        }
        else {
            //diffuse shading
            //divide color by 1 - probDiffuse
            //this probably shouldn't ever happen if our material is valid
            //because this means that m.specular.color != 0 but hasReflective & hasRefractive == 0
            if (probDiffuse != 0.f) {
                sampleDiffuse(pathSegment, normal, m, 1.f - probDiffuse);
            }
        }
        pathSegment.ray.origin = intersect + pathSegment.ray.direction * 0.001f;
    }
}
