#include "hip/hip_runtime.h"
#include "interactions.h"

__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine &rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng)); // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}

__host__ __device__ float reflectance(float cosine, float refraction_index) {
    // Use Schlick's approximation for reflectance.
    float r0 = (1.0 - refraction_index) / (1.0 + refraction_index);
    r0 = r0 * r0;
    return r0 + (1.0 - r0) * glm::pow((1.0 - cosine), 5.0);
}

__host__ __device__ glm::vec3 getColorFromTexture(glm::vec2 uv, const Texture& texture) {
    int x = static_cast<int>(uv.x * texture.width) % texture.width;
    int y = static_cast<int>((1.f - uv.y) * texture.height) % texture.height;
    int index = (y * texture.width + x) * texture.channel;
    float r = texture.imgData[index] / 255.0f;
    float g = texture.imgData[index + 1] / 255.0f;
    float b = texture.imgData[index + 2] / 255.0f;
    return glm::vec3(r, g, b);
}

__host__ __device__ void scatterRay(
    PathSegment & pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    glm::vec2 uv,
    bool outside,
    const Material &m,
    glm::vec3 materialColor,
    thrust::default_random_engine &rng)
{
    // TODO: implement this.
    // A basic implementation of pure-diffuse shading will just call the
    // calculateRandomDirectionInHemisphere defined above.
    thrust::uniform_real_distribution<float> u01(0, 1);

    glm::vec3 newDirection;
    glm::vec3 currDir = normalize(pathSegment.ray.direction);
    glm::vec3 n = normalize(normal);

    pathSegment.ray.origin = intersect + 0.01f * n;

    if (m.hasReflective) {
        glm::vec3 reflectedDir = glm::reflect(currDir, n);
        // Perfect specular reflection
        newDirection = reflectedDir;
        pathSegment.color *= materialColor;
    }
    else if (m.hasRefractive) {
        float rand_f = u01(rng);
        
        float cos_theta = glm::min(1.f, dot(-currDir, n));
        float schlick_factor = reflectance(cos_theta, m.indexOfRefraction);
        if (schlick_factor >= rand_f) {
            newDirection = normalize(glm::reflect(currDir, n));
            pathSegment.ray.origin = intersect + 0.01f * newDirection;
            pathSegment.color *= materialColor;
        }
        else {
            float eta = m.indexOfRefraction;

            if (outside) {
                eta = 1.0 / m.indexOfRefraction;
            }
            glm::vec3 refracted = glm::refract(currDir, n, eta);
            newDirection = normalize(refracted);
            pathSegment.ray.origin = intersect + 0.01f * newDirection;
            pathSegment.color *= materialColor;
        }
        
    }
    else {
        newDirection = calculateRandomDirectionInHemisphere(n, rng);
        pathSegment.color *= materialColor;
    }
    pathSegment.ray.direction = normalize(newDirection);

    pathSegment.remainingBounces--;
}
