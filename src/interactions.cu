#include "hip/hip_runtime.h"
#include "interactions.h"

__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine &rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng));      // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal + cos(around) * over * perpendicularDirection1 + sin(around) * over * perpendicularDirection2;
}

__host__ __device__ Sample sampleLight(
    glm::vec3 viewPoint,
    const Geom &geom,
    const Material *materials,
    thrust::default_random_engine &rng)
{
    Material material = materials[geom.materialid];
    if (geom.type == SQUARE)
    {
        thrust::uniform_real_distribution<float> uSquareSide(-0.5, 0.5);
        glm::vec2 squarePoint = glm::vec2(uSquareSide(rng), uSquareSide(rng));
        glm::vec3 lightPoint = multiplyMV(geom.transform, glm::vec4(squarePoint, 0, 1));
        glm::vec3 r = lightPoint - viewPoint;
        glm::vec3 incomingDirection = glm::normalize(r);
        float pdfdA = 1.f / (geom.scale.x * geom.scale.y);
        float rSquare = dot(r, r);
        glm::vec3 normal = glm::normalize(multiplyMV(geom.invTranspose, glm::vec4(0, 0, 1, 0)));
        float pdfdw = rSquare / dot(-incomingDirection, normal) * pdfdA;

        return Sample{
            .incomingDirection = incomingDirection,
            .value = material.color * material.emittance,
            .pdf = pdfdw,
            .delta = false,
        };
    }
    else if (geom.type == CUBE)
    {
        // Picks randomly from 3 closest sides of the cube
        glm::vec3 viewpointObj = multiplyMV(geom.inverseTransform, glm::vec4(viewPoint, 1));
        thrust::uniform_real_distribution<float> uSquareSide(-0.5, 0.5);

        thrust::uniform_int_distribution<int> u02(0, 2);
        int axis = u02(rng);

        glm::vec3 normalObj = glm::vec3();
        glm::vec3 lightPointObj;
        float pdfdA = 1.f / (geom.scale.x * geom.scale.y * geom.scale.z);
        if (axis == 0) {
            normalObj.x = glm::sign(viewpointObj.x);
            lightPointObj = 0.5f * normalObj + glm::vec3(0, uSquareSide(rng), uSquareSide(rng));
            pdfdA *= geom.scale.x;
        } else if (axis == 1) {
            normalObj.y = glm::sign(viewpointObj.y);
            lightPointObj = 0.5f * normalObj + glm::vec3(uSquareSide(rng), 0, uSquareSide(rng));
            pdfdA *= geom.scale.y;
        } else {
            normalObj.z = glm::sign(viewpointObj.z);
            lightPointObj = 0.5f * normalObj + glm::vec3(uSquareSide(rng), uSquareSide(rng), 0);
            pdfdA *= geom.scale.z;
        }

        glm::vec3 lightPoint = multiplyMV(geom.transform, glm::vec4(lightPointObj, 1));
        glm::vec3 r = lightPoint - viewPoint;
        glm::vec3 incomingDirection = glm::normalize(r);

        float rSquare = dot(r, r);
        glm::vec3 normal = glm::normalize(multiplyMV(geom.invTranspose, glm::vec4(normalObj, 0)));

        float cosTheta = dot(-incomingDirection, normalObj);
        float pdfdw = rSquare / std::abs(cosTheta) * pdfdA;

        return Sample {
            .incomingDirection = incomingDirection,
            .value = (cosTheta > 0) ? (material.color * material.emittance) : glm::vec3(0, 1, 1),
            .pdf = std::abs(pdfdw) / 3,
            .delta = false};
    }
    else if (geom.type == SPHERE)
    {
        // Assumption: sampling happens from outside the sphere (will be the case for most geom)
        thrust::uniform_real_distribution<float> u01(0, 1);
        glm::vec3 originObj = multiplyMV(geom.inverseTransform, glm::vec4(viewPoint, 1));
        glm::vec3 lightPointObj = calculateRandomDirectionInHemisphere(originObj, rng);
        glm::vec3 lightPoint = multiplyMV(geom.transform, glm::vec4(lightPointObj, 1));
        glm::vec3 normal = glm::normalize(multiplyMV(geom.invTranspose, glm::vec4(lightPointObj, 0)));
        glm::vec3 r = lightPoint - viewPoint;
        glm::vec3 incomingDirection = glm::normalize(r);

        // TODO: Check math here for uneven scaling?
        float pdfdA = 2 * PI * geom.scale.x * geom.scale.y * geom.scale.z / glm::length(lightPointObj / geom.scale);
        float rSquare = dot(r, r);
        float pdfdw = rSquare / dot(-incomingDirection, normal) * pdfdA;

        return Sample{
            .incomingDirection = incomingDirection,
            .value = material.color * material.emittance,
            .pdf = pdfdw,
            .delta = false};
    }

    return Sample();
}

__host__ __device__ Sample sampleBsdf(
    const Material &material,
    glm::vec3 normal,
    glm::vec3 outgoingDirection,
    thrust::default_random_engine &rng)
{
    if (material.hasReflective)
    {
        return Sample{
            .incomingDirection = glm::reflect(outgoingDirection, normal),
            .value = material.specular.color,
            .pdf = 1.f,
            .delta = true};
    }
    return Sample{
        .incomingDirection = calculateRandomDirectionInHemisphere(normal, rng),
        .value = material.color / PI,
        .pdf = 1 / PI,
        .delta = false};
}

__host__ __device__ glm::vec3 getBsdf(const Material &material, glm::vec3 normal, glm::vec3 incomingDirection, glm::vec3 outgoingDirection)
{
    if (material.hasReflective)
    {
        return glm::vec3(0);
    }

    return material.color / PI;
}

__host__ __device__ void scatterRay(
    PathSegment &pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    const Material &m,
    thrust::default_random_engine &rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    Sample sampleBsdfImportance = sampleBsdf(m, normal, pathSegment.ray.direction, rng);

    const float clipping_offset = 0.01f;
    pathSegment.ray.direction = sampleBsdfImportance.incomingDirection;
    pathSegment.ray.origin = intersect + sampleBsdfImportance.incomingDirection * clipping_offset;
    pathSegment.throughput *= sampleBsdfImportance.value / sampleBsdfImportance.pdf;
}
