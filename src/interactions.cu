#include "hip/hip_runtime.h"
#include "interactions.h"

#define TEST_DIFFUSE 0
#define USE_DIFFUSE_TEXTURE 1
#define USE_NORMAL_TEXTURE 1
#define TEST_NORMAL_MAP 0

__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine &rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng)); // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}

__host__ __device__ glm::vec3 textureSample(const Texture* texture, glm::vec2 uv) {
    int x = (int)(uv.x * texture->width);
    int y = (int)((1.0f - uv.y) * texture->height);
    int index = y * texture->width + x;

    glm::vec3 texColor = texture->data[index];
    return texColor;
}

__host__ __device__ void scatterRay(
    PathSegment & pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    glm::vec3 tangent,
    glm::vec3 bitangent,
    glm::vec2 uv,
    //glm::mat3 TBN,
    const Material &m,
    thrust::default_random_engine &rng)
{
    // TODO: implement this.
    // A basic implementation of pure-diffuse shading will just call the
    // calculateRandomDirectionInHemisphere defined above.

    // Set the new ray origin and direction to the intersection point
	glm::vec3 newOrigin = intersect + EPSILON * normal;   
    glm::vec3 newDirection = normalize(calculateRandomDirectionInHemisphere(normal, rng));

    float pdf = 1.0;

#if !TEST_DIFFUSE
    // Diffuse shading (Lambertian reflection)
    if (!m.hasReflective && !m.hasRefractive && !m.hasDiffuseTexture) {
        // Simple diffuse scattering
        pathSegment.ray.direction = newDirection;

		float cosTheta = glm::abs(glm::dot(normal, newDirection));
		pdf = cosTheta / PI;

        // throughput
        if (pdf > EPSILON) {
            pathSegment.color *= m.color;
        }
    }
    else if (m.hasReflective) {
        // Reflective materials: Specular reflection
        pathSegment.ray.direction = glm::reflect(pathSegment.ray.direction, normal);
		// calculate specular reflection color
        
        // Adjust the color for reflective materials using the specular component
        pathSegment.color *= m.color;
    }
    else if (m.hasRefractive) {
        thrust::uniform_real_distribution<float> u01(0, 1);
        float rand = u01(rng);
        float cosTheta = glm::dot(normal, pathSegment.ray.direction);
        float eta = (cosTheta > 0) ? (m.indexOfRefraction / 1.0f) : (1.0f / m.indexOfRefraction);
        glm::vec3 refractDirection = glm::refract(pathSegment.ray.direction, normal, eta);

        // Adjust normal direction and cosTheta for refraction calculations
        if (cosTheta < 0) {
            cosTheta = -cosTheta; //entering the medium
        }
        else {
            //normal = -normal; // Flip the normal
        }

        // Calculate Fresnel reflectance using Schlick's approximation
        float R0 = pow((1.0f - m.indexOfRefraction) / (1.0f + m.indexOfRefraction), 2);
        float reflectance = R0 + (1 - R0) * pow(1 - cosTheta, 5);

        // Check if the refraction results in total internal reflection
        /*if (glm::length(refractDirection) == 0) {
            reflectance = 1.0; 
        }*/

        if (rand < reflectance) {
            // Reflect
            newDirection = glm::reflect(pathSegment.ray.direction, normal);
        }
        else {
            // Refract
            newDirection = refractDirection;
        }
		newOrigin = intersect - EPSILON * normal;
        pathSegment.color *= m.color;      
    }




    // texture
#if USE_DIFFUSE_TEXTURE
    if (m.hasDiffuseTexture) {
        glm::vec3 textureColor = textureSample(m.diffuseTexture, uv);
        // gamma correction to texture color
        textureColor = glm::pow(textureColor, glm::vec3(1.0f / 2.2f));
        pathSegment.color *= textureColor;
    }
#endif
#if USE_NORMAL_TEXTURE
    if (m.hasNormalTexture) {
        glm::vec3 sampledNormal = textureSample(m.normalTexture, uv);
        sampledNormal = 2.0f * sampledNormal - glm::vec3(1.0f); // [0,1] to [-1, 1]
		glm::mat3 TBN = glm::mat3(tangent, bitangent, normal);
        normal = normalize(TBN * sampledNormal);
        newOrigin = intersect;
        newDirection = normalize(calculateRandomDirectionInHemisphere(normal, rng));
#if TEST_NORMAL_MAP
        pathSegment.color = normal;
#endif
    }
#endif

#else   
    //test diffuse
	

	float cosTheta = glm::dot(normal, newDirection);
	pdf = cosTheta * INV_PI;

	// Multiply the path color by the material color (throughput)
	if (pdf > EPSILON) {
		pathSegment.color *= m.color; // why not divide by PI?
	}
    else {
		pathSegment.color = glm::vec3(0.0f);
        return;
    }

#endif

	// UPDATE the path segment with the new ray origin and direction
    pathSegment.ray.origin = newOrigin;
    pathSegment.ray.direction = newDirection;
    pathSegment.remainingBounces--;
}
