#include "hip/hip_runtime.h"
#include "interactions.h"

__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine &rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng)); // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}

__host__ __device__  glm::vec3 samplePointOnLight(const Geom& geom, thrust::default_random_engine& rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    if (geom.type == SPHERE)
    {
        float theta = 2 * PI * u01(rng);
        float phi = acos(2 * u01(rng) - 1);
        float r = geom.scale.x; // Assuming uniform scale for sphere

        glm::vec3 localPoint(
            r * sin(phi) * cos(theta),
            r * sin(phi) * sin(theta),
            r * cos(phi)
        );

        return geom.translation + glm::vec3(geom.transform * glm::vec4(localPoint, 1.0f));
    }
    else if (geom.type == CUBE)
    {
        // Randomly choose one of the 6 faces
        int face = int(u01(rng) * 6);
        float u = u01(rng);
        float v = u01(rng);

        glm::vec3 localPoint;
        switch (face)
        {
        case 0: localPoint = glm::vec3(-0.5f, u - 0.5f, v - 0.5f); break; // Left
        case 1: localPoint = glm::vec3(0.5f, u - 0.5f, v - 0.5f); break; // Right
        case 2: localPoint = glm::vec3(u - 0.5f, -0.5f, v - 0.5f); break; // Bottom
        case 3: localPoint = glm::vec3(u - 0.5f, 0.5f, v - 0.5f); break; // Top
        case 4: localPoint = glm::vec3(u - 0.5f, v - 0.5f, -0.5f); break; // Back
        case 5: localPoint = glm::vec3(u - 0.5f, v - 0.5f, 0.5f); break; // Front
        }

        return geom.translation + glm::vec3(geom.transform * glm::vec4(localPoint, 1.0f));
    }

    // Default case (shouldn't happen)
    return geom.translation;
}


__host__ __device__ void scatterRay(
    PathSegment& pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    const Material& m,
    thrust::default_random_engine& rng,
    Geom* geoms,
    int geoms_size,
    Material* materials)
{
    if (pathSegment.remainingBounces == 0)
        return;

    normal = glm::normalize(normal);
    glm::vec3 viewDir = -pathSegment.ray.direction;

    thrust::uniform_real_distribution<float> u01(0, 1);
    float rand = u01(rng);

    glm::vec3 newDirection;
    glm::vec3 indirectLight(0.0f);
    glm::vec3 directLight(0.0f);

    if (m.plasticSpecular > 0.0f)
    {
        float F0 = pow((m.indexOfRefraction - 1) / (m.indexOfRefraction + 1), 2);
        float cosTheta = glm::max(glm::dot(normal, viewDir), 0.0f);
        float fresnel = F0 + (1.0f - F0) * pow(1.0f - cosTheta, 5.0f);

        if (rand < fresnel)
        {
            // Specular reflection
            newDirection = glm::reflect(-viewDir, normal);

            if (m.roughness > 0.0f)
            {
                glm::vec3 randomDir = calculateRandomDirectionInHemisphere(normal, rng);
                newDirection = glm::normalize(glm::mix(newDirection, randomDir, m.roughness));
            }

            pathSegment.color *= m.specular.color;
        }
        else
        {
            // Diffuse reflection
            newDirection = calculateRandomDirectionInHemisphere(normal, rng);
            pathSegment.color *= m.color;
        }
    }
    else if (m.hasReflective > 0.0f)
    {
        // Metallic reflection
        newDirection = glm::reflect(-viewDir, normal);

        if (m.roughness > 0.0f)
        {
            glm::vec3 randomDir = calculateRandomDirectionInHemisphere(normal, rng);
            newDirection = glm::normalize(glm::mix(newDirection, randomDir, m.roughness));
        }

        pathSegment.color *= glm::mix(m.specular.color, m.color * glm::dot(newDirection, normal), m.metallic);
    }
    else if (m.hasRefractive > 0.0f)
    {
        // Refractive surface
        float n1 = 1.0f; // Assume air as the surrounding medium
        float n2 = m.indexOfRefraction;
        float eta = n1 / n2;
        float cosThetaI = glm::dot(-pathSegment.ray.direction, normal);
        float k = 1.0f - eta * eta * (1.0f - cosThetaI * cosThetaI);

        if (k >= 0.0f)
        {
            newDirection = eta * pathSegment.ray.direction + (eta * cosThetaI - sqrtf(k)) * normal;
            pathSegment.color *= m.color;
        }
        else
        {
            newDirection = glm::reflect(pathSegment.ray.direction, normal);
            pathSegment.color *= m.specular.color;
        }
    }
    else
    {
        // Diffuse reflection
        newDirection = calculateRandomDirectionInHemisphere(normal, rng);
        pathSegment.color *= m.color;
    }

    indirectLight = pathSegment.color;

#if DIRECT_LIGHTING
    // Compute direct lighting
    for (int i = 0; i < geoms_size; ++i)
    {
        if (materials[geoms[i].materialid].emittance > 0.0f)
        {
            glm::vec3 lightPoint = samplePointOnLight(geoms[i], rng);
            glm::vec3 lightDir = glm::normalize(lightPoint - intersect);
            float lightDistance = glm::length(lightPoint - intersect);

            Ray shadowRay;
            shadowRay.origin = intersect + normal * 0.001f; // Offset to avoid self-intersection
            shadowRay.direction = lightDir;
            bool occluded = false;

            for (int j = 0; j < geoms_size; ++j)
            {
                if (j != i) // Don't check intersection with the light itself
                {
                    float t;
                    glm::vec3 tmpIntersect, tmpNormal;
                    bool tmpOutside;
                    if (geoms[j].type == CUBE)
                        t = boxIntersectionTest(geoms[j], shadowRay, tmpIntersect, tmpNormal, tmpOutside);
                    else if (geoms[j].type == SPHERE)
                        t = sphereIntersectionTest(geoms[j], shadowRay, tmpIntersect, tmpNormal, tmpOutside);

                    if (t > 0 && t < lightDistance)
                    {
                        occluded = true;
                        break;
                    }
                }
            }

            if (!occluded)
            {
                float cosTheta = glm::max(glm::dot(normal, lightDir), 0.0f);
                glm::vec3 lightColor = materials[geoms[i].materialid].color;
                float lightIntensity = materials[geoms[i].materialid].emittance;

                glm::vec3 brdf = m.color / PI;

                directLight += (brdf * lightColor * lightIntensity * cosTheta) / (lightDistance * lightDistance);
            }
        }
    }
#endif
    pathSegment.color = indirectLight + directLight;
    pathSegment.ray.origin = intersect + newDirection * 0.001f;
    pathSegment.ray.direction = glm::normalize(newDirection);
    pathSegment.remainingBounces--;
}
